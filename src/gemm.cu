#include "hip/hip_runtime.h"
#include <cutf/cublas.hpp>
#include "config.hpp"
#include "split.hpp"
#include "utils.hpp"
#include "handle.hpp"

namespace {
template <class T>
std::size_t split_core(
		void* const split_ptr,
		const mtk::ozimmu::operation_t op,
		const std::size_t m,
		const std::size_t n,
		const T* const src_ptr, const std::size_t ld,
		const std::vector<mtk::ozimmu::data_t> data_type_list,
		const mtk::ozimmu::detail::matrix_t matrix,
		const T* const two_to_alpha_ptr,
		hipStream_t hip_stream
		) {
	const auto num_split = data_type_list.size() - 1;
	std::size_t offset = 0;

	if (num_split <= 1) {
		// Do nothing
	} else {
		OZIMMU_NOT_IMPLEMENTED;
	}

	return offset;
}

template <class T>
void split_AB_int8(
		mtk::ozimmu::handle_t handle,
		const mtk::ozimmu::operation_t op_A,
		const mtk::ozimmu::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const T* const a_ptr, const std::size_t lda,
		double* const a_max_exp_ptr,
		std::int8_t* const working_a_ptr,
		const T* const b_ptr, const std::size_t ldb,
		double* const b_max_exp_ptr,
		std::int8_t* const working_b_ptr,
		const unsigned num_split,
		const unsigned bits_per_int8
		) {
	handle->profiler.start_timer_sync("split_A");
	mtk::ozimmu::split_int8<T>(
			working_a_ptr,
			a_max_exp_ptr,
			m, k,
			a_ptr, lda,
			op_A,
			mtk::ozimmu::detail::matrix_A,
			num_split,
			bits_per_int8,
			handle->hip_stream
			);
	handle->profiler.stop_timer_sync("split_A");

	handle->profiler.start_timer_sync("split_B");
	mtk::ozimmu::split_int8<T>(
			working_b_ptr,
			b_max_exp_ptr,
			k, n,
			b_ptr, ldb,
			op_B,
			mtk::ozimmu::detail::matrix_B,
			num_split,
			bits_per_int8,
			handle->hip_stream
			);
	handle->profiler.stop_timer_sync("split_B");
}

hipDataType to_cudaDataType_t(
		const mtk::ozimmu::data_t d
		) {
	switch (d) {
	case mtk::ozimmu::fp32:
		return HIP_R_32F;
	case mtk::ozimmu::fp16:
		return HIP_R_16F;
	default:
		break;
	}
	OZIMMU_NOT_IMPLEMENTED;
	return HIP_R_32F;
}

hipblasOperation_t to_cublasOperation_t(
		const mtk::ozimmu::operation_t op
		) {
	switch (op) {
	case mtk::ozimmu::op_n:
		return HIPBLAS_OP_N;
	case mtk::ozimmu::op_t:
		return HIPBLAS_OP_T;
	default:
		break;
	}
	OZIMMU_NOT_IMPLEMENTED;
	return HIPBLAS_OP_N;
}

__global__ void accumulate_in_f64_kernel(
		double* const f64_ptr,
		const std::int32_t* i32_ptr,
		const std::size_t length,
		const double scale
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= length) {
		return;
	}

	f64_ptr[tid] += static_cast<double>(static_cast<std::int64_t>(i32_ptr[tid]) << 32) * scale;
}

void accumulate_in_f64(
		double* const f64_ptr,
		const std::int32_t* i32_ptr,
		const std::size_t length,
		const std::int32_t mantissa_rshift,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;
	const auto scale = cutf::experimental::fp::reinterpret_as_fp(static_cast<std::uint64_t>((cutf::experimental::fp::get_bias<double>() - mantissa_rshift)) << cutf::experimental::fp::get_mantissa_size<double>());
	accumulate_in_f64_kernel
		<<<(length + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
				f64_ptr,
				i32_ptr,
				length,
				scale
			);
}

template <class T>
__global__ void init_accumulator_buffer_kernel(
		T* const dp_ptr,
		const std::size_t length
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= length) {
		return;
	}

	dp_ptr[tid] = 0;
}

template <class T>
void init_accumulator_buffer(
		T* const dp_ptr,
		const std::size_t length,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;
	init_accumulator_buffer_kernel<T>
		<<<(length + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
				dp_ptr,
				length
			);
}

__global__ void axby_kernel(
		const std::size_t m,
		const std::size_t n,
		const double a,
		const double* const x_ptr,
		const double b,
		double* const y_ptr,
		const std::size_t ldy,
		const double* const a_max_exp_ptr,
		const double* const b_max_exp_ptr
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= m * n) {
		return;
	}

	const auto mi = tid % m;
	const auto ni = tid / m;

	const auto memory_index = ni * ldy + mi;

	const auto x = x_ptr[tid] / (1l << 44) * a_max_exp_ptr[mi] * b_max_exp_ptr[ni];

	if (b != 0) {
		y_ptr[memory_index] = a * x + b * y_ptr[memory_index];
	} else {
		y_ptr[memory_index] = a * x;
	}
}

void axby(
		const std::size_t m,
		const std::size_t n,
		const double a,
		const double* const x_ptr,
		const double b,
		double* const y_ptr,
		const std::size_t ldy,
		const double* const a_max_exp_ptr,
		const double* const b_max_exp_ptr,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;
	axby_kernel
		<<<(m * n + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
				m, n,
				a,
				x_ptr,
				b,
				y_ptr, ldy,
				a_max_exp_ptr,
				b_max_exp_ptr
			);
}

__global__ void axy_complex_kernel(
		const std::size_t m,
		const std::size_t n,
		const hipDoubleComplex a,
		const double* const x_ptr,
		hipDoubleComplex* const y_ptr,
		const std::size_t ldy,
		const double* const a_max_exp_ptr,
		const double* const b_max_exp_ptr
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= m * n) {
		return;
	}

	const auto mi = tid % m;
	const auto ni = tid / m;

	const auto memory_index = ni * ldy + mi;

	const auto x = x_ptr[tid] / (1l << 44) * a_max_exp_ptr[mi] * b_max_exp_ptr[ni];

	auto y = y_ptr[memory_index];

	y.x = a.x * x + y.x;
	y.y = a.y * x + y.y;

	y_ptr[memory_index] = y;
}


void axy_complex(
		const std::size_t m,
		const std::size_t n,
		const hipDoubleComplex a,
		const double* const x_ptr,
		hipDoubleComplex* const y_ptr,
		const std::size_t ldy,
		const double* const a_max_exp_ptr,
		const double* const b_max_exp_ptr,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;
	axy_complex_kernel
		<<<(m * n + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
				m, n,
				a,
				x_ptr,
				y_ptr, ldy,
				a_max_exp_ptr,
				b_max_exp_ptr
			);
}

template <bool is_beta_zero>
__global__ void init_c_complex_kernel(
		const std::size_t m,
		const std::size_t n,
		hipDoubleComplex* const c_ptr,
		const std::size_t ldc,
		const hipDoubleComplex beta
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= m * n) {
		return;
	}

	const auto mi = tid % m;
	const auto ni = tid / m;

	const auto memory_index = ni * ldc + mi;

	if (is_beta_zero) {
		c_ptr[memory_index] = make_hipDoubleComplex(0, 0);
	} else {
		auto c = c_ptr[memory_index];
		c.x = c.x * beta.x - c.y * beta.y;
		c.y = c.y * beta.x + c.x * beta.y;

		c_ptr[memory_index] = c;
	}
}

void init_c_complex(
		const std::size_t m,
		const std::size_t n,
		hipDoubleComplex* const c_ptr,
		const std::size_t ldc,
		const hipDoubleComplex beta,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;

	if (beta.x == 0 && beta.y == 0) {
		init_c_complex_kernel<true>
			<<<(m * n + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
					m, n,
					c_ptr,
					ldc,
					beta
					);
	} else {
		init_c_complex_kernel<false>
			<<<(m * n + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
					m, n,
					c_ptr,
					ldc,
					beta
					);
	}
}

hipblasStatus_t cublasGemmEx_org(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n, int k,
		const void *alpha, const void *A,
		hipDataType Atype, int lda, const void *B,
		hipDataType Btype, int ldb, const void *beta,
		void *C, hipDataType Ctype, int ldc,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
	const std::string cublas_library_name = "libcublas.so";
	const std::string cublas_function_name = "hipblasGemmEx";
	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = ozIMMU_get_function_pointer(
			cublas_library_name.c_str(),
			cublas_function_name.c_str()
			);

	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);

	return res;
}

void gemm_core(
		mtk::ozimmu::handle_t handle,
		const mtk::ozimmu::operation_t op_A,
		const mtk::ozimmu::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const void* const a_ptr, const std::size_t lda, const mtk::ozimmu::data_t type_a,
		const void* const b_ptr, const std::size_t ldb, const mtk::ozimmu::data_t type_b,
		void* const c_ptr,
		const mtk::ozimmu::detail::gemm_pair_config_t& gemm_pair_config,
		const mtk::ozimmu::compute_mode_t compute_mode,
		const void* const a_working_memory_ptr,
		const void* const b_working_memory_ptr
		) {
	const auto gemm_mode = gemm_pair_config.gemm_mode;
	const auto split_config = mtk::ozimmu::detail::get_split_config(compute_mode);
	const auto lda_r = gemm_pair_config.A_id == 0 ? lda : k;
	const auto ldb_r = gemm_pair_config.B_id == 0 ? ldb : k;

	std::size_t A_working_ptr_offset = 0;
	for (unsigned i = 0; i < gemm_pair_config.A_id; i++) {
		const auto t = split_config.matrix_A_split_types[i];
		A_working_ptr_offset += m * k * mtk::ozimmu::get_data_size_in_byte(t);
	}

	std::size_t B_working_ptr_offset = 0;
	for (unsigned i = 0; i < gemm_pair_config.B_id; i++) {
		const auto t = split_config.matrix_B_split_types[i];
		B_working_ptr_offset += k * n * mtk::ozimmu::get_data_size_in_byte(t);
	}

	const void* const a_working_ptr = reinterpret_cast<const std::uint8_t*>(a_working_memory_ptr) + A_working_ptr_offset;
	const void* const b_working_ptr = reinterpret_cast<const std::uint8_t*>(b_working_memory_ptr) + B_working_ptr_offset;

	const void* const a_ptr_r = gemm_pair_config.A_id == 0 ? a_ptr : a_working_ptr;
	const void* const b_ptr_r = gemm_pair_config.B_id == 0 ? b_ptr : b_working_ptr;
	void* const c_ptr_r = c_ptr;

	const float alpha_r = 1, beta_r = 0;

	const auto profile_label = mtk::ozimmu::detail::gemm_mode_str(gemm_mode);
	handle->profiler.start_timer_sync(profile_label);
	switch (gemm_mode) {
	case mtk::ozimmu::detail::cublas_dgemm:
		{
			const double alpha_dp = 1, beta_dp = 0;
			const auto op_A_r = gemm_pair_config.A_id == 0 ? to_cublasOperation_t(op_A) : HIPBLAS_OP_T;
			const auto op_B_r = gemm_pair_config.B_id == 0 ? to_cublasOperation_t(op_B) : HIPBLAS_OP_N;

			const auto cublas_algorithm = HIPBLAS_GEMM_DEFAULT;

			const auto cublas_compute_mode = HIPBLAS_COMPUTE_64F;

			CUTF_CHECK_ERROR(cublasGemmEx_org(
						handle->cublas_handle,
						op_A_r,
						op_B_r,
						m, n, k,
						&alpha_dp,
						a_ptr_r, HIP_R_64F, lda_r,
						b_ptr_r, HIP_R_64F, ldb_r,
						&beta_dp,
						c_ptr_r, HIP_R_64F, m,
						cublas_compute_mode,
						cublas_algorithm
						));
		}
		break;
	case mtk::ozimmu::detail::cublas_sgemm:
	case mtk::ozimmu::detail::cublas_bf16:
	case mtk::ozimmu::detail::cublas_tf32:
	case mtk::ozimmu::detail::cublas_fp16:
		{
			const auto op_A_r = gemm_pair_config.A_id == 0 ? to_cublasOperation_t(op_A) : HIPBLAS_OP_T;
			const auto op_B_r = gemm_pair_config.B_id == 0 ? to_cublasOperation_t(op_B) : HIPBLAS_OP_N;
			const auto type_A_r = gemm_pair_config.A_id == 0 ? type_a : split_config.matrix_A_split_types[gemm_pair_config.A_id];
			const auto type_B_r = gemm_pair_config.B_id == 0 ? type_b : split_config.matrix_B_split_types[gemm_pair_config.B_id];

			const auto cublas_algorithm = gemm_mode == mtk::ozimmu::detail::cublas_sgemm ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP;

			auto cublas_compute_mode = HIPBLAS_COMPUTE_32F;
			if (gemm_mode == mtk::ozimmu::detail::cublas_bf16) cublas_compute_mode = HIPBLAS_COMPUTE_32F_FAST_16BF;
			else if (gemm_mode == mtk::ozimmu::detail::cublas_fp16) cublas_compute_mode = HIPBLAS_COMPUTE_32F_FAST_16F;
			else if (gemm_mode == mtk::ozimmu::detail::cublas_tf32) cublas_compute_mode = HIPBLAS_COMPUTE_32F_FAST_TF32;

			CUTF_CHECK_ERROR(cublasGemmEx_org(
						handle->cublas_handle,
						op_A_r,
						op_B_r,
						m, n, k,
						&alpha_r,
						a_ptr_r, to_cudaDataType_t(type_A_r), lda_r,
						b_ptr_r, to_cudaDataType_t(type_B_r), ldb_r,
						&beta_r,
						c_ptr_r, HIP_R_32F, m,
						cublas_compute_mode,
						cublas_algorithm
						));
		}
		break;
	case mtk::ozimmu::detail::int8tc:
		{
			const int alpha_i = 1, beta_i = 0;
			const auto op_A_r = gemm_pair_config.A_id == 0 ? to_cublasOperation_t(op_A) : HIPBLAS_OP_T;
			const auto op_B_r = gemm_pair_config.B_id == 0 ? to_cublasOperation_t(op_B) : HIPBLAS_OP_N;

			CUTF_CHECK_ERROR_M(cublasGemmEx_org(
						handle->cublas_handle,
						op_A_r,
						op_B_r,
						m, n, k,
						&alpha_i,
						a_ptr_r, HIP_R_8I, lda_r,
						b_ptr_r, HIP_R_8I, ldb_r,
						&beta_i,
						c_ptr_r, HIP_R_32I, m,
						HIPBLAS_COMPUTE_32I,
						CUBLAS_GEMM_DEFAULT_TENSOR_OP
						), ("GemmEx(int8)-m" + std::to_string(m) + "-n" + std::to_string(n) + "-k" + std::to_string(k)));
		}
		break;
	default:
		OZIMMU_NOT_IMPLEMENTED;
	}
	handle->profiler.stop_timer_sync(profile_label);
}

template <class T>
int gemm_int8(
		mtk::ozimmu::handle_t handle,
		const mtk::ozimmu::operation_t op_A,
		const mtk::ozimmu::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const T* alpha,
		const T* const a_ptr, const std::size_t lda,
		const T* const b_ptr, const std::size_t ldb,
		const T* beta,
		T* const c_ptr, std::size_t ldc,
		const mtk::ozimmu::compute_mode_t compute_mode
		);

template <>
int gemm_int8<double>(
		mtk::ozimmu::handle_t handle,
		const mtk::ozimmu::operation_t op_A,
		const mtk::ozimmu::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const double* alpha,
		const double* const a_ptr, const std::size_t lda,
		const double* const b_ptr, const std::size_t ldb,
		const double* beta,
		double* const c_ptr, std::size_t ldc,
		const mtk::ozimmu::compute_mode_t compute_mode
		) {
	const unsigned num_split = mtk::ozimmu::detail::get_split_config(compute_mode).matrix_A_split_types.size() - 1;
	const int32_t bits_per_int8 = mtk::ozimmu::get_bits_per_int8(k);

	std::int32_t* const c_i32_ptr = reinterpret_cast<std::int32_t*>(handle->working_memory_ptr);
	double* const c_f64_ptr = reinterpret_cast<double*>(c_i32_ptr + m * n);
	double* const a_max_exp_ptr = reinterpret_cast<double*>(c_f64_ptr + m * n);
	double* const b_max_exp_ptr = a_max_exp_ptr + m;
	void* const working_memory_ptr = b_max_exp_ptr + n;

	init_accumulator_buffer(
			c_f64_ptr,
			m * n,
			handle->hip_stream
			);

	split_AB_int8<double>(
			handle,
			op_A,
			op_B,
			m, n, k, a_ptr, lda,
			a_max_exp_ptr,
			reinterpret_cast<std::int8_t*>(working_memory_ptr),
			b_ptr, ldb,
			b_max_exp_ptr,
			reinterpret_cast<std::int8_t*>(working_memory_ptr) + m * k * num_split,
			num_split,
			bits_per_int8
			);

	std::size_t A_working_memory_size = mtk::ozimmu::detail::calculate_working_memory_size(m, k, compute_mode, mtk::ozimmu::detail::matrix_A, mtk::ozimmu::real);

	const auto& gemm_pair_config_list = mtk::ozimmu::detail::get_split_config(compute_mode).gemm_pair_config_list;
	for (const auto& gemm_pair_config : gemm_pair_config_list) {
		gemm_core(
				handle,
				op_A, op_B,
				m, n, k,
				a_ptr, lda, mtk::ozimmu::fp64,
				b_ptr, ldb, mtk::ozimmu::fp64,
				c_i32_ptr,
				gemm_pair_config,
				compute_mode,
				working_memory_ptr,
				reinterpret_cast<std::uint8_t*>(working_memory_ptr) + A_working_memory_size
				);
		handle->profiler.start_timer_sync("accumulate_in_f64");
		accumulate_in_f64(
				c_f64_ptr,
				c_i32_ptr,
				m * n,
				bits_per_int8 * (gemm_pair_config.A_id + gemm_pair_config.B_id - 2) - (7 /*bitlen(int8)-1*/ - bits_per_int8) * 2, // The `(7 - bits_per_int8) * 2` term is required because the mantissa `bits_per_int8` bits are stored in the low `bits_per_int8` bits of an int8.
				handle->hip_stream
				);
		handle->profiler.stop_timer_sync("accumulate_in_f64");
	}
	handle->profiler.start_timer_sync("copy_result");
	axby(
			m, n,
			*alpha,
			c_f64_ptr,
			*beta,
			c_ptr, ldc,
			a_max_exp_ptr,
			b_max_exp_ptr,
			handle->hip_stream
			);
	handle->profiler.stop_timer_sync("copy_result");

	return 0;
}

template <>
int gemm_int8<hipDoubleComplex>(
		mtk::ozimmu::handle_t handle,
		const mtk::ozimmu::operation_t op_A,
		const mtk::ozimmu::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const hipDoubleComplex* alpha,
		const hipDoubleComplex* const a_ptr, const std::size_t lda,
		const hipDoubleComplex* const b_ptr, const std::size_t ldb,
		const hipDoubleComplex* beta,
		hipDoubleComplex* const c_ptr, std::size_t ldc,
		const mtk::ozimmu::compute_mode_t compute_mode
		) {
	using real_t = double;
	const unsigned num_split = mtk::ozimmu::detail::get_split_config(compute_mode).matrix_A_split_types.size() - 1;
	const int32_t bits_per_int8 = mtk::ozimmu::get_bits_per_int8(k);
	const auto& gemm_pair_config_list = mtk::ozimmu::detail::get_split_config(compute_mode).gemm_pair_config_list;

	std::int32_t* const c_i32_ptr = reinterpret_cast<std::int32_t*>(handle->working_memory_ptr);
	double* const tmp_f64_ptr = reinterpret_cast<double*>(c_i32_ptr + m * n);
	double* const a_real_max_exp_ptr = reinterpret_cast<double*>(tmp_f64_ptr + m * n);
	double* const a_imag_max_exp_ptr = a_real_max_exp_ptr + m;
	double* const b_real_max_exp_ptr = a_imag_max_exp_ptr + m;
	double* const b_imag_max_exp_ptr = b_real_max_exp_ptr + n;
	void* const working_memory_ptr = b_imag_max_exp_ptr + n;

	const double* a_max_exp_ptr_list[] = {
		a_real_max_exp_ptr,
		a_imag_max_exp_ptr
	};
	const std::int8_t* a_int8_working_memory_ptr_list[] = {
		reinterpret_cast<const std::int8_t*>(working_memory_ptr),
		reinterpret_cast<const std::int8_t*>(working_memory_ptr) + mtk::ozimmu::detail::calculate_working_memory_size(m, k, compute_mode, mtk::ozimmu::detail::matrix_A, mtk::ozimmu::real),
	};

	const double* b_max_exp_ptr_list[] = {
		b_real_max_exp_ptr,
		b_imag_max_exp_ptr
	};
	const std::int8_t* b_int8_working_memory_ptr_list[] = {
		a_int8_working_memory_ptr_list[0] + mtk::ozimmu::detail::calculate_working_memory_size(m, k, compute_mode, mtk::ozimmu::detail::matrix_A, mtk::ozimmu::complx),
		a_int8_working_memory_ptr_list[0] + mtk::ozimmu::detail::calculate_working_memory_size(m, k, compute_mode, mtk::ozimmu::detail::matrix_A, mtk::ozimmu::complx) + mtk::ozimmu::detail::calculate_working_memory_size(k, n, compute_mode, mtk::ozimmu::detail::matrix_B, mtk::ozimmu::real),
	};

	split_AB_int8<hipDoubleComplex>(
			handle,
			op_A,
			op_B,
			m, n, k,
			a_ptr, lda,
			a_real_max_exp_ptr,
			reinterpret_cast<std::int8_t*>(working_memory_ptr),
			b_ptr, ldb,
			b_real_max_exp_ptr,
			reinterpret_cast<std::int8_t*>(working_memory_ptr) + m * k * num_split * 2,
			num_split,
			bits_per_int8
			);

	// Init C
	init_c_complex(
			m, n,
			c_ptr, ldc,
			*beta,
			handle->hip_stream
			);

	for (const auto p : std::vector<std::pair<unsigned, unsigned>>{{1, 1}, {0, 0}, {1, 0}, {0, 1}}) {
		init_accumulator_buffer(
				tmp_f64_ptr,
				m * n,
				handle->hip_stream
				);
		for (const auto& gemm_pair_config : gemm_pair_config_list) {
			gemm_core(
					handle,
					op_A, op_B,
					m, n, k,
					a_ptr, lda, mtk::ozimmu::fp64,
					b_ptr, ldb, mtk::ozimmu::fp64,
					c_i32_ptr,
					gemm_pair_config,
					compute_mode,
					a_int8_working_memory_ptr_list[p.first],
					b_int8_working_memory_ptr_list[p.second]
					);
			handle->profiler.start_timer_sync("accumulate_in_f64");
			accumulate_in_f64(
					tmp_f64_ptr,
					c_i32_ptr,
					m * n,
					bits_per_int8 * (gemm_pair_config.A_id + gemm_pair_config.B_id - 2) - (7 /*bitlen(int8)-1*/ - bits_per_int8) * 2, // The `(7 - bits_per_int8) * 2` term is required because the mantissa `bits_per_int8` bits are stored in the low `bits_per_int8` bits of an int8.
					handle->hip_stream
					);
			handle->profiler.stop_timer_sync("accumulate_in_f64");
		}

		real_t axpy_alpha_real = 0;
		real_t axpy_alpha_imag = 0;
		if (p.first == 0 && p.second == 0) {
			axpy_alpha_real = alpha->x;
			axpy_alpha_imag = alpha->y;
		} else if (p.first == 1 && p.second == 1) {
			axpy_alpha_real = -alpha->x;
			axpy_alpha_imag = -alpha->y;
		} else {
			axpy_alpha_real = -alpha->y;
			axpy_alpha_imag = alpha->x;
		}
		handle->profiler.start_timer_sync("copy_result");
		axy_complex(
				m, n,
				make_hipDoubleComplex(axpy_alpha_real, axpy_alpha_imag),
				tmp_f64_ptr,
				c_ptr, ldc,
				a_max_exp_ptr_list[p.first],
				b_max_exp_ptr_list[p.second],
				handle->hip_stream
				);
		handle->profiler.stop_timer_sync("copy_result");
	}

	return 0;
}
} // unnamed namespace

int mtk::ozimmu::gemm(
		mtk::ozimmu::handle_t handle,
		const mtk::ozimmu::operation_t op_A,
		const mtk::ozimmu::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const void* alpha,
		const void* const a_ptr, const std::size_t lda,
		const void* const b_ptr, const std::size_t ldb,
		const void* beta,
		void* const c_ptr, std::size_t ldc,
		const mtk::ozimmu::compute_mode_t compute_mode,
		const mtk::ozimmu::element_kind_t element_kind
		) {
	// Arguments validation
	int arg_errors = 0;
	arg_errors += check_gemm_shape(op_A, m, k, lda, "A");
	arg_errors += check_gemm_shape(op_B, k, n, ldb, "B");
	arg_errors += check_gemm_shape(mtk::ozimmu::op_n, m, n, ldc, "C");
	if (arg_errors) {
		return 1;
	}

	mtk::ozimmu::data_t input_type;
	switch (compute_mode) {
		case mtk::ozimmu::sgemm:
			input_type = mtk::ozimmu::fp32;
			break;
		case mtk::ozimmu::dgemm:
		case mtk::ozimmu::fp64_int8_3:
		case mtk::ozimmu::fp64_int8_4:
		case mtk::ozimmu::fp64_int8_5:
		case mtk::ozimmu::fp64_int8_6:
		case mtk::ozimmu::fp64_int8_7:
		case mtk::ozimmu::fp64_int8_8:
		case mtk::ozimmu::fp64_int8_9:
		case mtk::ozimmu::fp64_int8_10:
		case mtk::ozimmu::fp64_int8_11:
		case mtk::ozimmu::fp64_int8_12:
		case mtk::ozimmu::fp64_int8_13:
		case mtk::ozimmu::fp64_int8_14:
		case mtk::ozimmu::fp64_int8_15:
		case mtk::ozimmu::fp64_int8_16:
		case mtk::ozimmu::fp64_int8_17:
		case mtk::ozimmu::fp64_int8_18:
		case mtk::ozimmu::fp64_int8_auto:
			input_type = mtk::ozimmu::fp64;
			break;
		default:
			OZIMMU_NOT_IMPLEMENTED;
	}

	gemm_list_t gemm_list = {
		std::tuple<std::size_t, std::size_t, std::size_t, mtk::ozimmu::element_kind_t, mtk::ozimmu::compute_mode_t>{m, n, k, element_kind, compute_mode}
	};
	mtk::ozimmu::reallocate_working_memory(handle, gemm_list);

	if (input_type == mtk::ozimmu::fp64) {
		if (
				compute_mode == mtk::ozimmu::fp64_int8_3  ||
				compute_mode == mtk::ozimmu::fp64_int8_4  ||
				compute_mode == mtk::ozimmu::fp64_int8_5  ||
				compute_mode == mtk::ozimmu::fp64_int8_6  ||
				compute_mode == mtk::ozimmu::fp64_int8_7  ||
				compute_mode == mtk::ozimmu::fp64_int8_8  ||
				compute_mode == mtk::ozimmu::fp64_int8_9  ||
				compute_mode == mtk::ozimmu::fp64_int8_10 ||
				compute_mode == mtk::ozimmu::fp64_int8_11 ||
				compute_mode == mtk::ozimmu::fp64_int8_12 ||
				compute_mode == mtk::ozimmu::fp64_int8_13 ||
				compute_mode == mtk::ozimmu::fp64_int8_14 ||
				compute_mode == mtk::ozimmu::fp64_int8_15 ||
				compute_mode == mtk::ozimmu::fp64_int8_16 ||
				compute_mode == mtk::ozimmu::fp64_int8_17 ||
				compute_mode == mtk::ozimmu::fp64_int8_18
				) {
			if (element_kind == mtk::ozimmu::real) {
				using T = double;
				gemm_int8(handle, op_A, op_B, m, n, k, reinterpret_cast<const T*>(alpha), reinterpret_cast<const T*>(a_ptr), lda, reinterpret_cast<const T*>(b_ptr), ldb, reinterpret_cast<const T*>(beta), reinterpret_cast<T*>(c_ptr), ldc, compute_mode);
			} else {
				using T = hipDoubleComplex;
				gemm_int8(handle, op_A, op_B, m, n, k, reinterpret_cast<const T*>(alpha), reinterpret_cast<const T*>(a_ptr), lda, reinterpret_cast<const T*>(b_ptr), ldb, reinterpret_cast<const T*>(beta), reinterpret_cast<T*>(c_ptr), ldc, compute_mode);
			}
		} else if (compute_mode == mtk::ozimmu::fp64_int8_auto) {
			const auto auto_mode = mtk::ozimmu::auto_mode_select(
						handle,
						op_A,
						op_B,
						m, n, k,
						a_ptr, lda,
						b_ptr, ldb,
						element_kind,
						handle->avg_mantissa_loss_threshold
					);
			ozIMMU_log("AUTO selected mode = " + mtk::ozimmu::get_compute_mode_name_str(auto_mode) + ", threshold average mantissa loss = " + std::to_string(handle->avg_mantissa_loss_threshold));
			return mtk::ozimmu::gemm(
					handle,
					op_A, op_B,
					m, n, k,
					alpha,
					a_ptr, lda,
					b_ptr, ldb,
					beta,
					c_ptr, ldc,
					auto_mode,
					element_kind
					);
		} else if (compute_mode == mtk::ozimmu::dgemm) {
			const auto dtype = element_kind == mtk::ozimmu::real ? HIP_R_64F : HIP_C_64F;
				cublasGemmEx_org(
						handle->cublas_handle,
						to_cublasOperation_t(op_A),
						to_cublasOperation_t(op_B),
						m, n, k,
						alpha,
						a_ptr, dtype, lda,
						b_ptr, dtype, ldb,
						beta,
						c_ptr, dtype, ldc,
						HIPBLAS_COMPUTE_64F,
						HIPBLAS_GEMM_DEFAULT
						);
		} else {
			OZIMMU_NOT_IMPLEMENTED;
		}
	} else {
		OZIMMU_NOT_IMPLEMENTED;
	}
	return 0;
}
