#include "hip/hip_runtime.h"
#include <cutf/cublas.hpp>
#include "config.hpp"
#include "split.hpp"
#include "utils.hpp"
#include "handle.hpp"

namespace {
template <class T>
std::size_t split_core(
		void* const split_ptr,
		const mtk::ozimma::operation_t op,
		const std::size_t m,
		const std::size_t n,
		const T* const src_ptr, const std::size_t ld,
		const std::vector<mtk::ozimma::data_t> data_type_list,
		const mtk::ozimma::detail::matrix_t matrix,
		const T* const two_to_alpha_ptr,
		hipStream_t hip_stream
		) {
	const auto num_split = data_type_list.size() - 1;
	std::size_t offset = 0;

	if (num_split <= 1) {
		// Do nothing
	} else {
		OZIMMA_NOT_IMPLEMENTED;
	}

	return offset;
}

template <class T>
void split_AB_int8(
		mtk::ozimma::handle_t handle,
		const mtk::ozimma::operation_t op_A,
		const mtk::ozimma::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const T* const a_ptr, const std::size_t lda,
		double* const a_max_exp_ptr,
		std::int8_t* const working_a_ptr,
		const T* const b_ptr, const std::size_t ldb,
		double* const b_max_exp_ptr,
		std::int8_t* const working_b_ptr,
		const unsigned num_split,
		const unsigned bits_per_int8
		) {
	handle->profiler.start_timer_sync("split_A");
	mtk::ozimma::split_int8<T>(
			working_a_ptr,
			a_max_exp_ptr,
			m, k,
			a_ptr, lda,
			op_A,
			mtk::ozimma::detail::matrix_A,
			num_split,
			bits_per_int8,
			handle->hip_stream
			);
	handle->profiler.stop_timer_sync("split_A");

	handle->profiler.start_timer_sync("split_B");
	mtk::ozimma::split_int8<T>(
			working_b_ptr,
			b_max_exp_ptr,
			k, n,
			b_ptr, ldb,
			op_B,
			mtk::ozimma::detail::matrix_B,
			num_split,
			bits_per_int8,
			handle->hip_stream
			);
	handle->profiler.stop_timer_sync("split_B");
}

hipDataType to_cudaDataType_t(
		const mtk::ozimma::data_t d
		) {
	switch (d) {
	case mtk::ozimma::fp32:
		return HIP_R_32F;
	case mtk::ozimma::fp16:
		return HIP_R_16F;
	default:
		break;
	}
	OZIMMA_NOT_IMPLEMENTED;
	return HIP_R_32F;
}

hipblasOperation_t to_cublasOperation_t(
		const mtk::ozimma::operation_t op
		) {
	switch (op) {
	case mtk::ozimma::op_n:
		return HIPBLAS_OP_N;
	case mtk::ozimma::op_t:
		return HIPBLAS_OP_T;
	default:
		break;
	}
	OZIMMA_NOT_IMPLEMENTED;
	return HIPBLAS_OP_N;
}

__global__ void accumulate_in_f64_kernel(
		double* const f64_ptr,
		const std::int32_t* i32_ptr,
		const std::size_t length,
		const double scale
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= length) {
		return;
	}

	f64_ptr[tid] += static_cast<double>(static_cast<std::int64_t>(i32_ptr[tid]) << 32) * scale;
}

void accumulate_in_f64(
		double* const f64_ptr,
		const std::int32_t* i32_ptr,
		const std::size_t length,
		const unsigned mantissa_rshift,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;
	const auto scale = cutf::experimental::fp::reinterpret_as_fp(static_cast<std::uint64_t>((cutf::experimental::fp::get_bias<double>() - mantissa_rshift)) << cutf::experimental::fp::get_mantissa_size<double>());
	accumulate_in_f64_kernel
		<<<(length + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
				f64_ptr,
				i32_ptr,
				length,
				scale
			);
}

template <class T>
__global__ void init_accumulator_buffer_kernel(
		T* const dp_ptr,
		const std::size_t length
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= length) {
		return;
	}

	dp_ptr[tid] = 0;
}

template <class T>
void init_accumulator_buffer(
		T* const dp_ptr,
		const std::size_t length,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;
	init_accumulator_buffer_kernel<T>
		<<<(length + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
				dp_ptr,
				length
			);
}

__global__ void axby_kernel(
		const std::size_t m,
		const std::size_t n,
		const double a,
		const double* const x_ptr,
		const double b,
		double* const y_ptr,
		const std::size_t ldy,
		const double* const a_max_exp_ptr,
		const double* const b_max_exp_ptr
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= m * n) {
		return;
	}

	const auto mi = tid % m;
	const auto ni = tid / m;

	const auto memory_index = ni * ldy + mi;

	const auto x = x_ptr[tid] / (1l << 44) * a_max_exp_ptr[mi] * b_max_exp_ptr[ni];

	if (b != 0) {
		y_ptr[memory_index] = a * x + b * y_ptr[memory_index];
	} else {
		y_ptr[memory_index] = a * x;
	}
}

void axby(
		const std::size_t m,
		const std::size_t n,
		const double a,
		const double* const x_ptr,
		const double b,
		double* const y_ptr,
		const std::size_t ldy,
		const double* const a_max_exp_ptr,
		const double* const b_max_exp_ptr,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;
	axby_kernel
		<<<(m * n + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
				m, n,
				a,
				x_ptr,
				b,
				y_ptr, ldy,
				a_max_exp_ptr,
				b_max_exp_ptr
			);
}

__global__ void axy_complex_kernel(
		const std::size_t m,
		const std::size_t n,
		const hipDoubleComplex a,
		const double* const x_ptr,
		hipDoubleComplex* const y_ptr,
		const std::size_t ldy,
		const double* const a_max_exp_ptr,
		const double* const b_max_exp_ptr
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= m * n) {
		return;
	}

	const auto mi = tid % m;
	const auto ni = tid / m;

	const auto memory_index = ni * ldy + mi;

	const auto x = x_ptr[tid] / (1l << 44) * a_max_exp_ptr[mi] * b_max_exp_ptr[ni];

	auto y = y_ptr[memory_index];

	y.x = a.x * x + y.x;
	y.y = a.y * x + y.y;

	y_ptr[memory_index] = y;
}


void axy_complex(
		const std::size_t m,
		const std::size_t n,
		const hipDoubleComplex a,
		const double* const x_ptr,
		hipDoubleComplex* const y_ptr,
		const std::size_t ldy,
		const double* const a_max_exp_ptr,
		const double* const b_max_exp_ptr,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;
	axy_complex_kernel
		<<<(m * n + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
				m, n,
				a,
				x_ptr,
				y_ptr, ldy,
				a_max_exp_ptr,
				b_max_exp_ptr
			);
}

template <bool is_beta_zero>
__global__ void init_c_complex_kernel(
		const std::size_t m,
		const std::size_t n,
		hipDoubleComplex* const c_ptr,
		const std::size_t ldc,
		const hipDoubleComplex beta
		) {
	const auto tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= m * n) {
		return;
	}

	const auto mi = tid % m;
	const auto ni = tid / m;

	const auto memory_index = ni * ldc + mi;

	if (is_beta_zero) {
		c_ptr[memory_index] = make_hipDoubleComplex(0, 0);
	} else {
		auto c = c_ptr[memory_index];
		c.x = c.x * beta.x - c.y * beta.y;
		c.y = c.y * beta.x + c.x * beta.y;

		c_ptr[memory_index] = c;
	}
}

void init_c_complex(
		const std::size_t m,
		const std::size_t n,
		hipDoubleComplex* const c_ptr,
		const std::size_t ldc,
		const hipDoubleComplex beta,
		hipStream_t hip_stream
		) {
	constexpr std::size_t block_size = 256;

	if (beta.x == 0 && beta.y == 0) {
		init_c_complex_kernel<true>
			<<<(m * n + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
					m, n,
					c_ptr,
					ldc,
					beta
					);
	} else {
		init_c_complex_kernel<false>
			<<<(m * n + block_size - 1) / block_size, block_size, 0, hip_stream>>>(
					m, n,
					c_ptr,
					ldc,
					beta
					);
	}
}

hipblasStatus_t cublasGemmEx_org(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n, int k,
		const void *alpha, const void *A,
		hipDataType Atype, int lda, const void *B,
		hipDataType Btype, int ldb, const void *beta,
		void *C, hipDataType Ctype, int ldc,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
	const std::string cublas_library_name = "libcublas.so";
	const std::string cublas_function_name = "hipblasGemmEx";
	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = ozIMMA_get_function_pointer(
			cublas_library_name.c_str(),
			cublas_function_name.c_str()
			);

	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);

	return res;
}

void gemm_core(
		mtk::ozimma::handle_t handle,
		const mtk::ozimma::operation_t op_A,
		const mtk::ozimma::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const void* const a_ptr, const std::size_t lda, const mtk::ozimma::data_t type_a,
		const void* const b_ptr, const std::size_t ldb, const mtk::ozimma::data_t type_b,
		void* const c_ptr,
		const mtk::ozimma::detail::gemm_pair_config_t& gemm_pair_config,
		const mtk::ozimma::compute_mode_t compute_mode,
		const void* const a_working_memory_ptr,
		const void* const b_working_memory_ptr
		) {
	const auto gemm_mode = gemm_pair_config.gemm_mode;
	const auto split_config = mtk::ozimma::detail::get_split_config(compute_mode);
	const auto lda_r = gemm_pair_config.A_id == 0 ? lda : k;
	const auto ldb_r = gemm_pair_config.B_id == 0 ? ldb : k;

	std::size_t A_working_ptr_offset = 0;
	for (unsigned i = 0; i < gemm_pair_config.A_id; i++) {
		const auto t = split_config.matrix_A_split_types[i];
		A_working_ptr_offset += m * k * mtk::ozimma::get_data_size_in_byte(t);
	}

	std::size_t B_working_ptr_offset = 0;
	for (unsigned i = 0; i < gemm_pair_config.B_id; i++) {
		const auto t = split_config.matrix_B_split_types[i];
		B_working_ptr_offset += k * n * mtk::ozimma::get_data_size_in_byte(t);
	}

	const void* const a_working_ptr = reinterpret_cast<const std::uint8_t*>(a_working_memory_ptr) + A_working_ptr_offset;
	const void* const b_working_ptr = reinterpret_cast<const std::uint8_t*>(b_working_memory_ptr) + B_working_ptr_offset;

	const void* const a_ptr_r = gemm_pair_config.A_id == 0 ? a_ptr : a_working_ptr;
	const void* const b_ptr_r = gemm_pair_config.B_id == 0 ? b_ptr : b_working_ptr;
	void* const c_ptr_r = c_ptr;

	const float alpha_r = 1, beta_r = 0;

	const auto profile_label = mtk::ozimma::detail::gemm_mode_str(gemm_mode);
	handle->profiler.start_timer_sync(profile_label);
	switch (gemm_mode) {
	case mtk::ozimma::detail::cublas_dgemm:
		{
			const double alpha_dp = 1, beta_dp = 0;
			const auto op_A_r = gemm_pair_config.A_id == 0 ? to_cublasOperation_t(op_A) : HIPBLAS_OP_T;
			const auto op_B_r = gemm_pair_config.B_id == 0 ? to_cublasOperation_t(op_B) : HIPBLAS_OP_N;

			const auto cublas_algorithm = HIPBLAS_GEMM_DEFAULT;

			const auto cublas_compute_mode = HIPBLAS_COMPUTE_64F;

			CUTF_CHECK_ERROR(cublasGemmEx_org(
						handle->cublas_handle,
						op_A_r,
						op_B_r,
						m, n, k,
						&alpha_dp,
						a_ptr_r, HIP_R_64F, lda_r,
						b_ptr_r, HIP_R_64F, ldb_r,
						&beta_dp,
						c_ptr_r, HIP_R_64F, m,
						cublas_compute_mode,
						cublas_algorithm
						));
		}
		break;
	case mtk::ozimma::detail::cublas_sgemm:
	case mtk::ozimma::detail::cublas_bf16:
	case mtk::ozimma::detail::cublas_tf32:
	case mtk::ozimma::detail::cublas_fp16:
		{
			const auto op_A_r = gemm_pair_config.A_id == 0 ? to_cublasOperation_t(op_A) : HIPBLAS_OP_T;
			const auto op_B_r = gemm_pair_config.B_id == 0 ? to_cublasOperation_t(op_B) : HIPBLAS_OP_N;
			const auto type_A_r = gemm_pair_config.A_id == 0 ? type_a : split_config.matrix_A_split_types[gemm_pair_config.A_id];
			const auto type_B_r = gemm_pair_config.B_id == 0 ? type_b : split_config.matrix_B_split_types[gemm_pair_config.B_id];

			const auto cublas_algorithm = gemm_mode == mtk::ozimma::detail::cublas_sgemm ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP;

			auto cublas_compute_mode = HIPBLAS_COMPUTE_32F;
			if (gemm_mode == mtk::ozimma::detail::cublas_bf16) cublas_compute_mode = HIPBLAS_COMPUTE_32F_FAST_16BF;
			else if (gemm_mode == mtk::ozimma::detail::cublas_fp16) cublas_compute_mode = HIPBLAS_COMPUTE_32F_FAST_16F;
			else if (gemm_mode == mtk::ozimma::detail::cublas_tf32) cublas_compute_mode = HIPBLAS_COMPUTE_32F_FAST_TF32;

			CUTF_CHECK_ERROR(cublasGemmEx_org(
						handle->cublas_handle,
						op_A_r,
						op_B_r,
						m, n, k,
						&alpha_r,
						a_ptr_r, to_cudaDataType_t(type_A_r), lda_r,
						b_ptr_r, to_cudaDataType_t(type_B_r), ldb_r,
						&beta_r,
						c_ptr_r, HIP_R_32F, m,
						cublas_compute_mode,
						cublas_algorithm
						));
		}
		break;
	case mtk::ozimma::detail::int8tc:
		{
			const int alpha_i = 1, beta_i = 0;
			const auto op_A_r = gemm_pair_config.A_id == 0 ? to_cublasOperation_t(op_A) : HIPBLAS_OP_T;
			const auto op_B_r = gemm_pair_config.B_id == 0 ? to_cublasOperation_t(op_B) : HIPBLAS_OP_N;

			CUTF_CHECK_ERROR_M(cublasGemmEx_org(
						handle->cublas_handle,
						op_A_r,
						op_B_r,
						m, n, k,
						&alpha_i,
						a_ptr_r, HIP_R_8I, lda_r,
						b_ptr_r, HIP_R_8I, ldb_r,
						&beta_i,
						c_ptr_r, HIP_R_32I, m,
						HIPBLAS_COMPUTE_32I,
						CUBLAS_GEMM_DEFAULT_TENSOR_OP
						), ("GemmEx(int8)-m" + std::to_string(m) + "-n" + std::to_string(n) + "-k" + std::to_string(k)));
		}
		break;
	default:
		OZIMMA_NOT_IMPLEMENTED;
	}
	handle->profiler.stop_timer_sync(profile_label);
}

template <class T>
int gemm_int8(
		mtk::ozimma::handle_t handle,
		const mtk::ozimma::operation_t op_A,
		const mtk::ozimma::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const T* alpha,
		const T* const a_ptr, const std::size_t lda,
		const T* const b_ptr, const std::size_t ldb,
		const T* beta,
		T* const c_ptr, std::size_t ldc,
		const mtk::ozimma::compute_mode_t compute_mode
		);

template <>
int gemm_int8<double>(
		mtk::ozimma::handle_t handle,
		const mtk::ozimma::operation_t op_A,
		const mtk::ozimma::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const double* alpha,
		const double* const a_ptr, const std::size_t lda,
		const double* const b_ptr, const std::size_t ldb,
		const double* beta,
		double* const c_ptr, std::size_t ldc,
		const mtk::ozimma::compute_mode_t compute_mode
		) {
	const unsigned num_split = mtk::ozimma::detail::get_split_config(compute_mode).matrix_A_split_types.size() - 1;
	const auto bits_per_int8 = std::min<unsigned>(7u, std::ceil((31 - std::log2(k) / 2.)));

	std::int32_t* const c_i32_ptr = reinterpret_cast<std::int32_t*>(handle->working_memory_ptr);
	double* const c_f64_ptr = reinterpret_cast<double*>(c_i32_ptr + m * n);
	double* const a_max_exp_ptr = reinterpret_cast<double*>(c_f64_ptr + m * n);
	double* const b_max_exp_ptr = a_max_exp_ptr + m;
	void* const working_memory_ptr = b_max_exp_ptr + n;

	init_accumulator_buffer(
			c_f64_ptr,
			m * n,
			handle->hip_stream
			);

	split_AB_int8<double>(
			handle,
			op_A,
			op_B,
			m, n, k, a_ptr, lda,
			a_max_exp_ptr,
			reinterpret_cast<std::int8_t*>(working_memory_ptr),
			b_ptr, ldb,
			b_max_exp_ptr,
			reinterpret_cast<std::int8_t*>(working_memory_ptr) + m * k * num_split,
			num_split,
			bits_per_int8
			);

	std::size_t A_working_memory_size = mtk::ozimma::detail::calculate_working_memory_size(m, n, compute_mode, mtk::ozimma::detail::matrix_A, mtk::ozimma::real);

	const auto& gemm_pair_config_list = mtk::ozimma::detail::get_split_config(compute_mode).gemm_pair_config_list;
	for (const auto& gemm_pair_config : gemm_pair_config_list) {
		gemm_core(
				handle,
				op_A, op_B,
				m, n, k,
				a_ptr, lda, mtk::ozimma::fp64,
				b_ptr, ldb, mtk::ozimma::fp64,
				c_i32_ptr,
				gemm_pair_config,
				compute_mode,
				working_memory_ptr,
				reinterpret_cast<std::uint8_t*>(working_memory_ptr) + A_working_memory_size
				);
		handle->profiler.start_timer_sync("accumulate_in_f64");
		accumulate_in_f64(
				c_f64_ptr,
				c_i32_ptr,
				m * n,
				bits_per_int8 * (gemm_pair_config.A_id + gemm_pair_config.B_id - 2),
				handle->hip_stream
				);
		handle->profiler.stop_timer_sync("accumulate_in_f64");
	}
	handle->profiler.start_timer_sync("copy_result");
	axby(
			m, n,
			*alpha,
			c_f64_ptr,
			*beta,
			c_ptr, ldc,
			a_max_exp_ptr,
			b_max_exp_ptr,
			handle->hip_stream
			);
	handle->profiler.stop_timer_sync("copy_result");

	return 0;
}

template <>
int gemm_int8<hipDoubleComplex>(
		mtk::ozimma::handle_t handle,
		const mtk::ozimma::operation_t op_A,
		const mtk::ozimma::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const hipDoubleComplex* alpha,
		const hipDoubleComplex* const a_ptr, const std::size_t lda,
		const hipDoubleComplex* const b_ptr, const std::size_t ldb,
		const hipDoubleComplex* beta,
		hipDoubleComplex* const c_ptr, std::size_t ldc,
		const mtk::ozimma::compute_mode_t compute_mode
		) {
	using real_t = double;
	const unsigned num_split = mtk::ozimma::detail::get_split_config(compute_mode).matrix_A_split_types.size() - 1;
	const auto bits_per_int8 = std::min<unsigned>(7u, std::ceil((31 - std::log2(k) / 2.)));
	const auto& gemm_pair_config_list = mtk::ozimma::detail::get_split_config(compute_mode).gemm_pair_config_list;

	std::int32_t* const c_i32_ptr = reinterpret_cast<std::int32_t*>(handle->working_memory_ptr);
	double* const tmp_f64_ptr = reinterpret_cast<double*>(c_i32_ptr + m * n);
	double* const a_real_max_exp_ptr = reinterpret_cast<double*>(tmp_f64_ptr + m * n);
	double* const a_imag_max_exp_ptr = a_real_max_exp_ptr + m;
	double* const b_real_max_exp_ptr = a_imag_max_exp_ptr + m;
	double* const b_imag_max_exp_ptr = b_real_max_exp_ptr + n;
	void* const working_memory_ptr = b_imag_max_exp_ptr + n;

	const double* a_max_exp_ptr_list[] = {
		a_real_max_exp_ptr,
		a_imag_max_exp_ptr
	};
	const std::int8_t* a_int8_working_memory_ptr_list[] = {
		reinterpret_cast<const std::int8_t*>(working_memory_ptr),
		reinterpret_cast<const std::int8_t*>(working_memory_ptr) + mtk::ozimma::detail::calculate_working_memory_size(m, k, compute_mode, mtk::ozimma::detail::matrix_A, mtk::ozimma::real),
	};

	const double* b_max_exp_ptr_list[] = {
		b_real_max_exp_ptr,
		b_imag_max_exp_ptr
	};
	const std::int8_t* b_int8_working_memory_ptr_list[] = {
		a_int8_working_memory_ptr_list[0] + mtk::ozimma::detail::calculate_working_memory_size(m, k, compute_mode, mtk::ozimma::detail::matrix_A, mtk::ozimma::complx),
		a_int8_working_memory_ptr_list[0] + mtk::ozimma::detail::calculate_working_memory_size(m, k, compute_mode, mtk::ozimma::detail::matrix_A, mtk::ozimma::complx) + mtk::ozimma::detail::calculate_working_memory_size(k, n, compute_mode, mtk::ozimma::detail::matrix_B, mtk::ozimma::real),
	};

	split_AB_int8<hipDoubleComplex>(
			handle,
			op_A,
			op_B,
			m, n, k,
			a_ptr, lda,
			a_real_max_exp_ptr,
			reinterpret_cast<std::int8_t*>(working_memory_ptr),
			b_ptr, ldb,
			b_real_max_exp_ptr,
			reinterpret_cast<std::int8_t*>(working_memory_ptr) + m * k * num_split * 2,
			num_split,
			bits_per_int8
			);

	// Init C
	init_c_complex(
			m, n,
			c_ptr, ldc,
			*beta,
			handle->hip_stream
			);

	for (const auto p : std::vector<std::pair<unsigned, unsigned>>{{1, 1}, {0, 0}, {1, 0}, {0, 1}}) {
		init_accumulator_buffer(
				tmp_f64_ptr,
				m * n,
				handle->hip_stream
				);
		for (const auto& gemm_pair_config : gemm_pair_config_list) {
			gemm_core(
					handle,
					op_A, op_B,
					m, n, k,
					a_ptr, lda, mtk::ozimma::fp64,
					b_ptr, ldb, mtk::ozimma::fp64,
					c_i32_ptr,
					gemm_pair_config,
					compute_mode,
					a_int8_working_memory_ptr_list[p.first],
					b_int8_working_memory_ptr_list[p.second]
					);
			handle->profiler.start_timer_sync("accumulate_in_f64");
			accumulate_in_f64(
					tmp_f64_ptr,
					c_i32_ptr,
					m * n,
					bits_per_int8 * (gemm_pair_config.A_id + gemm_pair_config.B_id - 2),
					handle->hip_stream
					);
			handle->profiler.stop_timer_sync("accumulate_in_f64");
		}
		
		real_t axpy_alpha_real = 0;
		real_t axpy_alpha_imag = 0;
		if (p.first == 0 && p.second == 0) {
			axpy_alpha_real = alpha->x;
			axpy_alpha_imag = alpha->y;
		} else if (p.first == 1 && p.second == 1) {
			axpy_alpha_real = -alpha->x;
			axpy_alpha_imag = -alpha->y;
		} else {
			axpy_alpha_real = -alpha->y;
			axpy_alpha_imag = alpha->x;
		}
		handle->profiler.start_timer_sync("copy_result");
		axy_complex(
				m, n,
				make_hipDoubleComplex(axpy_alpha_real, axpy_alpha_imag),
				tmp_f64_ptr,
				c_ptr, ldc,
				a_max_exp_ptr_list[p.first],
				b_max_exp_ptr_list[p.second],
				handle->hip_stream
				);
		handle->profiler.stop_timer_sync("copy_result");
	}

	return 0;
}
} // unnamed namespace

int mtk::ozimma::gemm(
		mtk::ozimma::handle_t handle,
		const mtk::ozimma::operation_t op_A,
		const mtk::ozimma::operation_t op_B,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const void* alpha,
		const void* const a_ptr, const std::size_t lda,
		const void* const b_ptr, const std::size_t ldb,
		const void* beta,
		void* const c_ptr, std::size_t ldc,
		const mtk::ozimma::compute_mode_t compute_mode,
		const mtk::ozimma::element_kind_t element_kind
		) {
	mtk::ozimma::data_t input_type;
	switch (compute_mode) {
	case mtk::ozimma::sgemm:
		input_type = mtk::ozimma::fp32;
		break;
	case mtk::ozimma::dgemm:
	case mtk::ozimma::fp64_int8_6:
	case mtk::ozimma::fp64_int8_7:
	case mtk::ozimma::fp64_int8_8:
	case mtk::ozimma::fp64_int8_9:
	case mtk::ozimma::fp64_int8_10:
	case mtk::ozimma::fp64_int8_11:
	case mtk::ozimma::fp64_int8_12:
	case mtk::ozimma::fp64_int8_13:
	case mtk::ozimma::fp64_int8_auto:
		input_type = mtk::ozimma::fp64;
		break;
	default:
		OZIMMA_NOT_IMPLEMENTED;
	}

	gemm_list_t gemm_list = {
		std::tuple<std::size_t, std::size_t, std::size_t, mtk::ozimma::element_kind_t, mtk::ozimma::compute_mode_t>{m, n, k, element_kind, compute_mode}
	};
	mtk::ozimma::reallocate_working_memory(handle, gemm_list);

	if (input_type == mtk::ozimma::fp64) {
		if (
				compute_mode == mtk::ozimma::fp64_int8_6  ||
				compute_mode == mtk::ozimma::fp64_int8_7  ||
				compute_mode == mtk::ozimma::fp64_int8_8  ||
				compute_mode == mtk::ozimma::fp64_int8_9  ||
				compute_mode == mtk::ozimma::fp64_int8_10 ||
				compute_mode == mtk::ozimma::fp64_int8_11 ||
				compute_mode == mtk::ozimma::fp64_int8_12 ||
				compute_mode == mtk::ozimma::fp64_int8_13
				) {
			if (element_kind == mtk::ozimma::real) {
				using T = double;
				gemm_int8(handle, op_A, op_B, m, n, k, reinterpret_cast<const T*>(alpha), reinterpret_cast<const T*>(a_ptr), lda, reinterpret_cast<const T*>(b_ptr), ldb, reinterpret_cast<const T*>(beta), reinterpret_cast<T*>(c_ptr), ldc, compute_mode);
			} else {
				using T = hipDoubleComplex;
				gemm_int8(handle, op_A, op_B, m, n, k, reinterpret_cast<const T*>(alpha), reinterpret_cast<const T*>(a_ptr), lda, reinterpret_cast<const T*>(b_ptr), ldb, reinterpret_cast<const T*>(beta), reinterpret_cast<T*>(c_ptr), ldc, compute_mode);
			}
		} else if (compute_mode == mtk::ozimma::fp64_int8_auto) {
			const auto auto_mode = mtk::ozimma::auto_mode_select(
						handle,
						op_A,
						op_B,
						m, n, k,
						a_ptr, lda,
						b_ptr, ldb,
						element_kind,
						1
					);
			ozIMMA_log("AUTO selected mode = " + mtk::ozimma::get_compute_mode_name_str(auto_mode));
			return mtk::ozimma::gemm(
					handle,
					op_A, op_B,
					m, n, k,
					alpha,
					a_ptr, lda,
					b_ptr, ldb,
					beta,
					c_ptr, ldc,
					auto_mode,
					element_kind
					);
		} else if (compute_mode == mtk::ozimma::dgemm) {
			const auto dtype = element_kind == mtk::ozimma::real ? HIP_R_64F : HIP_C_64F;
				cublasGemmEx_org(
						handle->cublas_handle,
						to_cublasOperation_t(op_A),
						to_cublasOperation_t(op_B),
						m, n, k,
						alpha,
						a_ptr, dtype, lda,
						b_ptr, dtype, ldb,
						beta,
						c_ptr, dtype, ldc,
						HIPBLAS_COMPUTE_64F,
						HIPBLAS_GEMM_DEFAULT
						);
		} else {
			OZIMMA_NOT_IMPLEMENTED;
		}
	} else {
		OZIMMA_NOT_IMPLEMENTED;
	}
	return 0;
}
