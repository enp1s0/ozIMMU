#include "hip/hip_runtime.h"
#include <cutf/device.hpp>
#include "handle.hpp"
#include "config.hpp"
#include "utils.hpp"

int mtk::ozimma::create(
		mtk::ozimma::handle_t *h,
		mtk::ozimma::malloc_mode_t mm
		) {
	ozIMMA_log("Initializing ozIMMA handle");
	auto handle = (*h = new mtk::ozimma::handle);
	// Initialize cuBLAS handler
	CUTF_CHECK_ERROR(cublasCreate_org(&(handle->cublas_handle)));

	handle->current_working_memory_size = 0;
	handle->working_memory_ptr = nullptr;
	handle->malloc_mode = mm;

	// Disable profiling by default
	mtk::ozimma::disable_profiling(*h);

	CUTF_CHECK_ERROR(hipMalloc(&(handle->d_mantissa_loss_counter_ptr), sizeof(unsigned long long int) * handle->mantissa_loss_counter_length));

	return 0;
}

int mtk::ozimma::destroy(
		mtk::ozimma::handle_t handle
		) {
	if (handle) {
		ozIMMA_log("Destroying ozIMMA handle");
		// Destroy cuBLAS handler
		CUTF_CHECK_ERROR(cublasDestroy_org(handle->cublas_handle));

		CUTF_CHECK_ERROR(hipFree(handle->working_memory_ptr));
		handle->working_memory_ptr = nullptr;

		CUTF_CHECK_ERROR(hipFree(handle->d_mantissa_loss_counter_ptr));
		handle->d_mantissa_loss_counter_ptr = nullptr;

		delete handle;
		handle = nullptr;
	}

	return 0;
}

void mtk::ozimma::set_cuda_stream(
		mtk::ozimma::handle_t handle,
		hipStream_t hip_stream
		) {
	// Set cuda stream to cuBLAS handler
	CUTF_CHECK_ERROR(hipblasSetStream(handle->cublas_handle, hip_stream));

	// Set ozimma handler
	handle->hip_stream = hip_stream;
}

std::size_t mtk::ozimma::reallocate_working_memory(
		mtk::ozimma::handle_t handle,
		const mtk::ozimma::gemm_list_t gemm_list
		) {
	std::size_t max_working_memory_size = 0;

	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		const auto element_kind = std::get<3>(gemm);
		const auto mode = std::get<4>(gemm);

		const auto working_memory_A = mtk::ozimma::detail::calculate_working_memory_size(m, k, mode, detail::matrix_A, element_kind);
		const auto working_memory_B = mtk::ozimma::detail::calculate_working_memory_size(k, n, mode, detail::matrix_B, element_kind);
		const auto working_memory_C_fp32 = m * n * mtk::ozimma::get_data_size_in_byte(fp32);
		const auto working_memory_C_fp64 = m * n * mtk::ozimma::get_data_size_in_byte(fp64) * (element_kind == mtk::ozimma::real ? 1 : 2);
		std::size_t etc = 0;
		if (
				mode == mtk::ozimma::fp64_int8_6  ||
				mode == mtk::ozimma::fp64_int8_7  ||
				mode == mtk::ozimma::fp64_int8_8  ||
				mode == mtk::ozimma::fp64_int8_9  ||
				mode == mtk::ozimma::fp64_int8_10 ||
				mode == mtk::ozimma::fp64_int8_11 ||
				mode == mtk::ozimma::fp64_int8_12 ||
				mode == mtk::ozimma::fp64_int8_13
			 ) {
			etc = (m + n) * mtk::ozimma::get_data_size_in_byte(fp64) * (element_kind == mtk::ozimma::real ? 1 : 2);
		}

		max_working_memory_size = std::max(
				max_working_memory_size,
				working_memory_A + working_memory_B + working_memory_C_fp32 + working_memory_C_fp64 + etc
				);
	}

	if (max_working_memory_size > handle->current_working_memory_size) {
		handle->current_working_memory_size = max_working_memory_size;

		ozIMMA_log("Reallocated moery : " + std::to_string(max_working_memory_size) + " B");

		if (handle->working_memory_ptr != nullptr) {
			if (handle->malloc_mode == mtk::ozimma::malloc_sync) {
				CUTF_CHECK_ERROR(hipFree(handle->working_memory_ptr));
			} else {
				CUTF_CHECK_ERROR(hipFreeAsync(handle->working_memory_ptr, handle->hip_stream));
			}
		}

		// Realloc
		if (handle->malloc_mode == mtk::ozimma::malloc_sync) {
			CUTF_CHECK_ERROR(hipMalloc(&(handle->working_memory_ptr), handle->current_working_memory_size));
		} else {
			CUTF_CHECK_ERROR(hipMallocAsync(&(handle->working_memory_ptr), handle->current_working_memory_size, handle->hip_stream));
		}

		return max_working_memory_size;
	}
	return 0;
}

std::string mtk::ozimma::get_compute_mode_name_str(
		const mtk::ozimma::compute_mode_t mode
		) {
	switch (mode) {
	case mtk::ozimma::sgemm:
		return "sgemm";
	case mtk::ozimma::dgemm:
		return "dgemm";
	case mtk::ozimma::fp64_int8_6:
		return "fp64_int8_6";
	case mtk::ozimma::fp64_int8_7:
		return "fp64_int8_7";
	case mtk::ozimma::fp64_int8_8:
		return "fp64_int8_8";
	case mtk::ozimma::fp64_int8_9:
		return "fp64_int8_9";
	case mtk::ozimma::fp64_int8_10:
		return "fp64_int8_10";
	case mtk::ozimma::fp64_int8_11:
		return "fp64_int8_11";
	case mtk::ozimma::fp64_int8_12:
		return "fp64_int8_12";
	case mtk::ozimma::fp64_int8_13:
		return "fp64_int8_13";
	case mtk::ozimma::fp64_int8_auto:
		return "fp64_int8_auto";
	default:
		break;
	}
	OZIMMA_NOT_IMPLEMENTED;
	return "";
}

mtk::ozimma::data_t mtk::ozimma::get_output_type(
		const mtk::ozimma::compute_mode_t compute_mode
		) {
	switch (compute_mode) {
	case mtk::ozimma::sgemm:
		return mtk::ozimma::fp32;

	case mtk::ozimma::fp64_int8_6:
	case mtk::ozimma::fp64_int8_7:
	case mtk::ozimma::fp64_int8_8:
	case mtk::ozimma::fp64_int8_9:
	case mtk::ozimma::fp64_int8_10:
	case mtk::ozimma::fp64_int8_11:
	case mtk::ozimma::fp64_int8_12:
	case mtk::ozimma::fp64_int8_13:
	case mtk::ozimma::fp64_int8_auto:
	case mtk::ozimma::dgemm:
		return mtk::ozimma::fp64;

	default:
		break;
	}
	OZIMMA_NOT_IMPLEMENTED;
	return mtk::ozimma::original;
}

std::size_t mtk::ozimma::get_data_size_in_byte(
		const mtk::ozimma::data_t d
		) {
	switch (d) {
	case mtk::ozimma::fp64:
		return 8;
	case mtk::ozimma::fp32:
		return 4;
	case mtk::ozimma::fp16:
		return 2;
	case mtk::ozimma::original:
		return 0;
	case mtk::ozimma::int8:
		return 1;
	default:
		break;
	}
	return 0;
}

void mtk::ozimma::enable_profiling(mtk::ozimma::handle_t handle) {
	handle->profiler.enable_measurement();
}

void mtk::ozimma::disable_profiling(mtk::ozimma::handle_t handle) {
	handle->profiler.disable_measurement();
}

void mtk::ozimma::print_profiler_result(mtk::ozimma::handle_t handle, const std::string tag, const bool csv) {
	if (!csv) {
		handle->profiler.print_result(tag);
	} else {
		handle->profiler.print_result_csv(tag);
	}
}

void mtk::ozimma::clear_profiler_result(mtk::ozimma::handle_t handle) {
	handle->profiler.clear();
}
