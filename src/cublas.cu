#include <cutf/cublas.hpp>
#include <ozimma/ozimma.hpp>
#include "culip.hpp"
#include "handle.hpp"
#include "utils.hpp"

#ifndef CUBLASAPI
#define CUBLASAPI
#endif

mtk::ozimma::gemm_list_t get_default_gemm_list() {
	return mtk::ozimma::gemm_list_t{
		std::tuple<std::size_t, std::size_t, std::size_t, mtk::ozimma::element_kind_t, mtk::ozimma::compute_mode_t>{1024, 1024, 1024, mtk::ozimma::real, mtk::ozimma::fp64_int8_9}
	};
}

mtk::ozimma::compute_mode_t get_compute_mode(
		const std::size_t m,
		const std::size_t n,
		const std::size_t k
		) {
	const char* env_name = "OZIMMA_COMPUTE_MODE";
	const char* env_val = getenv(env_name);

	std::vector<mtk::ozimma::compute_mode_t> supported_gemm_mode = {
		mtk::ozimma::sgemm,
		mtk::ozimma::dgemm,
		mtk::ozimma::fp64_int8_6,
		mtk::ozimma::fp64_int8_7,
		mtk::ozimma::fp64_int8_8,
		mtk::ozimma::fp64_int8_9,
		mtk::ozimma::fp64_int8_10,
		mtk::ozimma::fp64_int8_11,
		mtk::ozimma::fp64_int8_12,
		mtk::ozimma::fp64_int8_13,
		mtk::ozimma::fp64_int8_auto,
	};

	if (env_val != nullptr) {
		const std::string env_val_str = env_val;

		for (const auto mode : supported_gemm_mode) {
			if (mtk::ozimma::get_compute_mode_name_str(mode) == env_val_str) {
				return mode;
			}
		}
	}

	return mtk::ozimma::dgemm;
}

mtk::ozimma::operation_t op_cublas2oz(
		const hipblasOperation_t op
		) {
	if (op == HIPBLAS_OP_N) {
		return mtk::ozimma::op_n;
	} else {
		return mtk::ozimma::op_t;
	}
}

mtk::ozimma::handle_t* global_ozimma_handle = nullptr;

mtk::ozimma::handle_t& get_global_ozimma_handle() {
	if (global_ozimma_handle == nullptr) {
		mtk::ozimma::malloc_mode_t malloc_mode = mtk::ozimma::malloc_sync;
		ozTCECGEMM_run_if_env_defined(
				"OZIMMA_MALLOC_ASYNC",
				[&](){malloc_mode = mtk::ozimma::malloc_async;}
				);
		ozIMMA_log("Initializing ozIMMA handle...");
		global_ozimma_handle = new mtk::ozimma::handle_t;
		mtk::ozimma::create(global_ozimma_handle, malloc_mode);
		ozIMMA_log("Successfully initialized");
	}
	return *global_ozimma_handle;
}

std::string cublas_library_name = "libcublas.so";

hipblasStatus_t mtk::ozimma::cublasCreate_org(
		hipblasHandle_t* handle_ptr
		) {
	hipblasStatus_t (*func_ptr)(hipblasHandle_t*);
	*(void**)(&func_ptr)	= ozIMMA_get_function_pointer(cublas_library_name, "hipblasCreate");
	return (*func_ptr)(handle_ptr);
}

hipblasStatus_t mtk::ozimma::cublasDestroy_org(
		hipblasHandle_t cublas_handle
		) {
	hipblasStatus_t (*func_ptr)(hipblasHandle_t);
	*(void**)(&func_ptr)	= ozIMMA_get_function_pointer(cublas_library_name, "hipblasDestroy");
	return (*func_ptr)(cublas_handle);
}

// Hijacking functions
extern "C" {
CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCreate (hipblasHandle_t *handle) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	// Allocate memory
	const auto reallocated_size = mtk::ozimma::reallocate_working_memory(
			get_global_ozimma_handle(),
			get_default_gemm_list()
			);
	if (reallocated_size != 0) {
		ozIMMA_log("Reallocated moery : " + std::to_string(reallocated_size) + " B");
	}

	// Run the original function
	return mtk::ozimma::cublasCreate_org(handle);
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDestroy (hipblasHandle_t handle) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	if (global_ozimma_handle != nullptr) {
		ozIMMA_log("Destroying ozIMMA handle...");
		mtk::ozimma::destroy(
				get_global_ozimma_handle()
				);
		delete global_ozimma_handle;
		global_ozimma_handle = nullptr;
	}

	// Run the original function
	return mtk::ozimma::cublasDestroy_org(handle);
#endif
}

CUBLASAPI hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n, int k,
		const void *alpha, const void *A,
		hipDataType Atype, int lda, const void *B,
		hipDataType Btype, int ldb, const void *beta,
		void *C, hipDataType Ctype, int ldc,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	const auto compute_mode = get_compute_mode(m, n, k);
	if (compute_mode != mtk::ozimma::dgemm && m >= 1024 && n >= 1024 && k >= 1024 &&
			((Atype == HIP_R_64F && Btype == HIP_R_64F && Ctype == HIP_R_64F) || (Atype == HIP_C_64F && Btype == HIP_C_64F && Ctype == HIP_C_64F))
			) {
		const auto gemm_config = mtk::ozimma::gemm_list_t {
			std::tuple<std::size_t, std::size_t, std::size_t, mtk::ozimma::element_kind_t, mtk::ozimma::compute_mode_t>{m, n, k, mtk::ozimma::complx, compute_mode}
		};

		hipStream_t hip_stream;
		CUTF_CHECK_ERROR(hipblasGetStream(handle, &hip_stream));
		mtk::ozimma::set_cuda_stream(get_global_ozimma_handle(), hip_stream);

		mtk::ozimma::CULiP::profile_result profile_result;
		const auto profiling_flag = mtk::ozimma::CULiP::is_profiling_enabled();

		if (profiling_flag) {
			snprintf(profile_result.function_name, profile_result.function_name_length - 1,
					"%s%s-%s%s-m%d-n%d-k%d",
					(Atype == HIP_R_64F ? "D" : "Z"),
					mtk::ozimma::get_compute_mode_name_str(compute_mode).c_str(),
					mtk::ozimma::CULiP::get_cublasOperation_t_string(transa), mtk::ozimma::CULiP::get_cublasOperation_t_string(transb), m, n, k);
			mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}
		if (Atype == HIP_R_64F) {
			mtk::ozimma::gemm(
					get_global_ozimma_handle(),
					op_cublas2oz(transa),
					op_cublas2oz(transb),
					m, n, k,
					reinterpret_cast<const double*>(alpha),
					reinterpret_cast<const double*>(A), lda,
					reinterpret_cast<const double*>(B), ldb,
					reinterpret_cast<const double*>(beta),
					reinterpret_cast<double*>(C), ldc,
					compute_mode,
					mtk::ozimma::real
					);
		} else if (Atype == HIP_C_64F) {
			mtk::ozimma::gemm(
					get_global_ozimma_handle(),
					op_cublas2oz(transa),
					op_cublas2oz(transb),
					m, n, k,
					reinterpret_cast<const hipDoubleComplex*>(alpha),
					reinterpret_cast<const hipDoubleComplex*>(A), lda,
					reinterpret_cast<const hipDoubleComplex*>(B), ldb,
					reinterpret_cast<const hipDoubleComplex*>(beta),
					reinterpret_cast<hipDoubleComplex*>(C), ldc,
					compute_mode,
					mtk::ozimma::complx
					);
		}

		if (profiling_flag) {
			// Record end rimestamp
			mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::print_profile_result, (void*)&profile_result);
		}

		return HIPBLAS_STATUS_SUCCESS;
	}

	hipStream_t hip_stream;
	CUTF_CHECK_ERROR(hipblasGetStream(handle, &hip_stream));

	mtk::ozimma::CULiP::profile_result profile_result;
	const auto profiling_flag = mtk::ozimma::CULiP::is_profiling_enabled();

	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = ozIMMA_get_function_pointer(
			cublas_library_name.c_str(),
			__func__
			);

	if (profiling_flag) {
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%d-n%d-k%d", __func__, mtk::ozimma::CULiP::get_cublasOperation_t_string(transa), mtk::ozimma::CULiP::get_cublasOperation_t_string(transb), m, n, k);
		mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
	}

	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);

	if (profiling_flag) {
		// Record end rimestamp
		mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::print_profile_result, (void*)&profile_result);
	}

	return res;
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDgemm (hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const double *alpha,
		const double *A,
		int lda,
		const double *B,
		int ldb,
		const double *beta,
		double *C,
		int ldc) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	hipblasMath_t math_mode;
	CUTF_CHECK_ERROR(hipblasGetMathMode(handle, &math_mode));
	return hipblasGemmEx(
			handle,
			transa, transb,
			m, n, k,
			alpha,
			A, HIP_R_64F, lda,
			B, HIP_R_64F, ldb,
			beta,
			C, HIP_R_64F, ldc,
			HIPBLAS_COMPUTE_64F,
			(math_mode == HIPBLAS_DEFAULT_MATH ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP)
			);
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZgemm (hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const hipDoubleComplex *alpha,
		const hipDoubleComplex *A,
		int lda,
		const hipDoubleComplex *B,
		int ldb,
		const hipDoubleComplex *beta,
		hipDoubleComplex *C,
		int ldc) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	hipblasMath_t math_mode;
	CUTF_CHECK_ERROR(hipblasGetMathMode(handle, &math_mode));
	return hipblasGemmEx(
			handle,
			transa, transb,
			m, n, k,
			alpha,
			A, HIP_C_64F, lda,
			B, HIP_C_64F, ldb,
			beta,
			C, HIP_C_64F, ldc,
			HIPBLAS_COMPUTE_64F,
			(math_mode == HIPBLAS_DEFAULT_MATH ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP)
			);
#endif
}

CUBLASAPI hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n, int k,
		const void *alpha, const void *A,
		hipDataType Atype, int lda, long long int strideA, const void *B,
		hipDataType Btype, int ldb, long long int strideB, const void *beta,
		void *C, hipDataType Ctype, int ldc, long long int strideC,
		int batch_count,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	const auto compute_mode = get_compute_mode(m, n, k);
	if (compute_mode != mtk::ozimma::dgemm && m >= 1024 && n >= 1024 && k >= 1024 &&
			((Atype == HIP_R_64F && Btype == HIP_R_64F && Ctype == HIP_R_64F) || (Atype == HIP_C_64F && Btype == HIP_C_64F && Ctype == HIP_C_64F))
			) {
		const auto gemm_config = mtk::ozimma::gemm_list_t {
			std::tuple<std::size_t, std::size_t, std::size_t, mtk::ozimma::element_kind_t, mtk::ozimma::compute_mode_t>{m, n, k, mtk::ozimma::complx, compute_mode}
		};

		hipStream_t hip_stream;
		CUTF_CHECK_ERROR(hipblasGetStream(handle, &hip_stream));
		mtk::ozimma::set_cuda_stream(get_global_ozimma_handle(), hip_stream);

		mtk::ozimma::CULiP::profile_result profile_result;
		const auto profiling_flag = mtk::ozimma::CULiP::is_profiling_enabled();

		if (profiling_flag) {
			snprintf(profile_result.function_name, profile_result.function_name_length - 1,
					"%s%s_stridedBatched-%s%s-m%d-n%d-k%d-batch_count%d",
					(Atype == HIP_R_64F ? "D" : "Z"),
					mtk::ozimma::get_compute_mode_name_str(compute_mode).c_str(),
					mtk::ozimma::CULiP::get_cublasOperation_t_string(transa), mtk::ozimma::CULiP::get_cublasOperation_t_string(transb), m, n, k, batch_count);
			mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}

		for (int batch_id = 0; batch_id < batch_count; batch_id++) {
			if (Atype == HIP_R_64F) {
				mtk::ozimma::gemm(
						get_global_ozimma_handle(),
						op_cublas2oz(transa),
						op_cublas2oz(transb),
						m, n, k,
						reinterpret_cast<const double*>(alpha),
						reinterpret_cast<const double*>(A) + strideA * batch_id, lda,
						reinterpret_cast<const double*>(B) + strideB * batch_id, ldb,
						reinterpret_cast<const double*>(beta),
						reinterpret_cast<double*>(C) + strideC * batch_id, ldc,
						compute_mode,
						mtk::ozimma::real
						);
			} else if (Atype == HIP_C_64F) {
				mtk::ozimma::gemm(
						get_global_ozimma_handle(),
						op_cublas2oz(transa),
						op_cublas2oz(transb),
						m, n, k,
						reinterpret_cast<const hipDoubleComplex*>(alpha),
						reinterpret_cast<const hipDoubleComplex*>(A) + strideA * batch_id, lda,
						reinterpret_cast<const hipDoubleComplex*>(B) + strideB * batch_id, ldb,
						reinterpret_cast<const hipDoubleComplex*>(beta),
						reinterpret_cast<hipDoubleComplex*>(C) + strideC * batch_id, ldc,
						compute_mode,
						mtk::ozimma::complx
						);
			}
		}

		if (profiling_flag) {
			// Record end rimestamp
			mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::print_profile_result, (void*)&profile_result);
		}

		return HIPBLAS_STATUS_SUCCESS;
	}

	hipStream_t hip_stream;
	CUTF_CHECK_ERROR(hipblasGetStream(handle, &hip_stream));

	mtk::ozimma::CULiP::profile_result profile_result;
	const auto profiling_flag = mtk::ozimma::CULiP::is_profiling_enabled();

	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, long long int, const void*, hipDataType, int, long long int, const void*, void*, hipDataType, int, long long int, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = ozIMMA_get_function_pointer(
			cublas_library_name.c_str(),
			__func__
			);

	if (profiling_flag) {
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%d-n%d-k%d", __func__, mtk::ozimma::CULiP::get_cublasOperation_t_string(transa), mtk::ozimma::CULiP::get_cublasOperation_t_string(transb), m, n, k);
		mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
	}

	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, strideA, B, Btype, ldb, strideB, beta, C, Ctype, ldc, strideC, batch_count, computeType, algo);

	if (profiling_flag) {
		// Record end rimestamp
		mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		mtk::ozimma::CULiP::launch_function(hip_stream, &mtk::ozimma::CULiP::print_profile_result, (void*)&profile_result);
	}

	return res;
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDgemmStridedBatched(hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const double *alpha,
		const double *A,
		int lda,
		long long int strideA,
		const double *B,
		int ldb,
		long long int strideB,
		const double *beta,
		double *C,
		int ldc,
		long long int strideC,
		int batch_count
		) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	hipblasMath_t math_mode;
	CUTF_CHECK_ERROR(hipblasGetMathMode(handle, &math_mode));
	return hipblasGemmStridedBatchedEx(
			handle,
			transa, transb,
			m, n, k,
			alpha,
			A, HIP_R_64F, lda, strideA,
			B, HIP_R_64F, ldb, strideB,
			beta,
			C, HIP_R_64F, ldc, strideC,
			batch_count,
			HIPBLAS_COMPUTE_64F,
			(math_mode == HIPBLAS_DEFAULT_MATH ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP)
			);
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZgemmStridedBatched(hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const hipDoubleComplex *alpha,
		const hipDoubleComplex *A,
		int lda,
		long long int strideA,
		const hipDoubleComplex *B,
		int ldb,
		long long int strideB,
		const hipDoubleComplex *beta,
		hipDoubleComplex *C,
		int ldc,
		long long int strideC,
		int batch_count
		) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	hipblasMath_t math_mode;
	CUTF_CHECK_ERROR(hipblasGetMathMode(handle, &math_mode));
	return hipblasGemmStridedBatchedEx(
			handle,
			transa, transb,
			m, n, k,
			alpha,
			A, HIP_C_64F, lda, strideA,
			B, HIP_C_64F, ldb, strideB,
			beta,
			C, HIP_C_64F, ldc, strideC,
			batch_count,
			HIPBLAS_COMPUTE_64F,
			(math_mode == HIPBLAS_DEFAULT_MATH ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP)
			);
#endif
}
} // extern "C"
