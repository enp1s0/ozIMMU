#include <cutf/cublas.hpp>
#include <ozimmu/ozimmu.hpp>
#include "culip.hpp"
#include "handle.hpp"
#include "utils.hpp"

#ifndef CUBLASAPI
#define CUBLASAPI
#endif

mtk::ozimmu::gemm_list_t get_default_gemm_list() {
	return mtk::ozimmu::gemm_list_t{
		std::tuple<std::size_t, std::size_t, std::size_t, mtk::ozimmu::element_kind_t, mtk::ozimmu::compute_mode_t>{1024, 1024, 1024, mtk::ozimmu::real, mtk::ozimmu::fp64_int8_9}
	};
}

mtk::ozimmu::compute_mode_t get_compute_mode(
		const std::size_t m,
		const std::size_t n,
		const std::size_t k
		) {
	const char* env_name = "OZIMMU_COMPUTE_MODE";
	const char* env_val = getenv(env_name);

	std::vector<mtk::ozimmu::compute_mode_t> supported_gemm_mode = {
		mtk::ozimmu::sgemm,
		mtk::ozimmu::dgemm,
		mtk::ozimmu::fp64_int8_3,
		mtk::ozimmu::fp64_int8_4,
		mtk::ozimmu::fp64_int8_5,
		mtk::ozimmu::fp64_int8_6,
		mtk::ozimmu::fp64_int8_7,
		mtk::ozimmu::fp64_int8_8,
		mtk::ozimmu::fp64_int8_9,
		mtk::ozimmu::fp64_int8_10,
		mtk::ozimmu::fp64_int8_11,
		mtk::ozimmu::fp64_int8_12,
		mtk::ozimmu::fp64_int8_13,
		mtk::ozimmu::fp64_int8_14,
		mtk::ozimmu::fp64_int8_15,
		mtk::ozimmu::fp64_int8_16,
		mtk::ozimmu::fp64_int8_17,
		mtk::ozimmu::fp64_int8_18,
		mtk::ozimmu::fp64_int8_auto,
	};

	if (env_val != nullptr) {
		const std::string env_val_str = env_val;

		for (const auto mode : supported_gemm_mode) {
			if (mtk::ozimmu::get_compute_mode_name_str(mode) == env_val_str) {
				return mode;
			}
		}
	}

	return mtk::ozimmu::dgemm;
}

mtk::ozimmu::operation_t op_cublas2oz(
		const hipblasOperation_t op
		) {
	if (op == HIPBLAS_OP_N) {
		return mtk::ozimmu::op_n;
	} else {
		return mtk::ozimmu::op_t;
	}
}

mtk::ozimmu::handle_t* global_ozimmu_handle = nullptr;

mtk::ozimmu::handle_t& get_global_ozimmu_handle() {
	if (global_ozimmu_handle == nullptr) {
		mtk::ozimmu::malloc_mode_t malloc_mode = mtk::ozimmu::malloc_sync;
		ozTCECGEMM_run_if_env_defined(
				"OZIMMU_MALLOC_ASYNC",
				[&](){malloc_mode = mtk::ozimmu::malloc_async;}
				);
		ozIMMU_log("Initializing ozIMMU handle...");
		global_ozimmu_handle = new mtk::ozimmu::handle_t;
		mtk::ozimmu::create(global_ozimmu_handle, malloc_mode);
		ozIMMU_log("Successfully initialized");
	}

	const auto threshold_env = "OZIMMU_AUTO_AVG_MANTISSA_LOSS_THRESHOLD";
	const auto threshold_env_ptr = getenv(threshold_env);
	if (threshold_env_ptr != nullptr) {
		try {
			mtk::ozimmu::set_auto_mantissa_loss_threashold(*global_ozimmu_handle, std::stod(threshold_env_ptr));
		} catch(const std::exception& e) {
			throw std::runtime_error(std::string("ERROR: ") + e.what() + " [" + threshold_env + " = " + std::string(threshold_env_ptr) + "]");
		}
	}

	return *global_ozimmu_handle;
}

std::string cublas_library_name = "libcublas.so";

hipblasStatus_t mtk::ozimmu::cublasCreate_org(
		hipblasHandle_t* handle_ptr
		) {
	hipblasStatus_t (*func_ptr)(hipblasHandle_t*);
	*(void**)(&func_ptr)	= ozIMMU_get_function_pointer(cublas_library_name, "hipblasCreate");
	return (*func_ptr)(handle_ptr);
}

hipblasStatus_t mtk::ozimmu::cublasDestroy_org(
		hipblasHandle_t cublas_handle
		) {
	hipblasStatus_t (*func_ptr)(hipblasHandle_t);
	*(void**)(&func_ptr)	= ozIMMU_get_function_pointer(cublas_library_name, "hipblasDestroy");
	return (*func_ptr)(cublas_handle);
}

// Hijacking functions
extern "C" {
CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasCreate (hipblasHandle_t *handle) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	// Allocate memory
	const auto reallocated_size = mtk::ozimmu::reallocate_working_memory(
			get_global_ozimmu_handle(),
			get_default_gemm_list()
			);

	// Run the original function
	return mtk::ozimmu::cublasCreate_org(handle);
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDestroy (hipblasHandle_t handle) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	if (global_ozimmu_handle != nullptr) {
		ozIMMU_log("Destroying ozIMMU handle...");
		mtk::ozimmu::destroy(
				get_global_ozimmu_handle()
				);
		delete global_ozimmu_handle;
		global_ozimmu_handle = nullptr;
	}

	// Run the original function
	return mtk::ozimmu::cublasDestroy_org(handle);
#endif
}

CUBLASAPI hipblasStatus_t hipblasGemmEx(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n, int k,
		const void *alpha, const void *A,
		hipDataType Atype, int lda, const void *B,
		hipDataType Btype, int ldb, const void *beta,
		void *C, hipDataType Ctype, int ldc,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	const auto compute_mode = get_compute_mode(m, n, k);
	if (compute_mode != mtk::ozimmu::dgemm && m >= 1024 && n >= 1024 && k >= 1024 &&
			((Atype == HIP_R_64F && Btype == HIP_R_64F && Ctype == HIP_R_64F) || (Atype == HIP_C_64F && Btype == HIP_C_64F && Ctype == HIP_C_64F))
			) {
		const auto gemm_config = mtk::ozimmu::gemm_list_t {
			std::tuple<std::size_t, std::size_t, std::size_t, mtk::ozimmu::element_kind_t, mtk::ozimmu::compute_mode_t>{m, n, k, mtk::ozimmu::complx, compute_mode}
		};

		hipStream_t hip_stream;
		CUTF_CHECK_ERROR(hipblasGetStream(handle, &hip_stream));
		mtk::ozimmu::set_cuda_stream(get_global_ozimmu_handle(), hip_stream);

		mtk::ozimmu::CULiP::profile_result profile_result;
		const auto profiling_flag = mtk::ozimmu::CULiP::is_profiling_enabled();

		if (profiling_flag) {
			snprintf(profile_result.function_name, profile_result.function_name_length - 1,
					"%s%s-%s%s-m%d-n%d-k%d",
					(Atype == HIP_R_64F ? "D" : "Z"),
					mtk::ozimmu::get_compute_mode_name_str(compute_mode).c_str(),
					mtk::ozimmu::CULiP::get_cublasOperation_t_string(transa), mtk::ozimmu::CULiP::get_cublasOperation_t_string(transb), m, n, k);
			mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}
		int error = 0;
		if (Atype == HIP_R_64F) {
			error = mtk::ozimmu::gemm(
					get_global_ozimmu_handle(),
					op_cublas2oz(transa),
					op_cublas2oz(transb),
					m, n, k,
					reinterpret_cast<const double*>(alpha),
					reinterpret_cast<const double*>(A), lda,
					reinterpret_cast<const double*>(B), ldb,
					reinterpret_cast<const double*>(beta),
					reinterpret_cast<double*>(C), ldc,
					compute_mode,
					mtk::ozimmu::real
					);
		} else if (Atype == HIP_C_64F) {
			error = mtk::ozimmu::gemm(
					get_global_ozimmu_handle(),
					op_cublas2oz(transa),
					op_cublas2oz(transb),
					m, n, k,
					reinterpret_cast<const hipDoubleComplex*>(alpha),
					reinterpret_cast<const hipDoubleComplex*>(A), lda,
					reinterpret_cast<const hipDoubleComplex*>(B), ldb,
					reinterpret_cast<const hipDoubleComplex*>(beta),
					reinterpret_cast<hipDoubleComplex*>(C), ldc,
					compute_mode,
					mtk::ozimmu::complx
					);
		}

		if (profiling_flag) {
			// Record end rimestamp
			mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::print_profile_result, (void*)&profile_result);
		}
		if (error) {
			return HIPBLAS_STATUS_INTERNAL_ERROR;
		}

		return HIPBLAS_STATUS_SUCCESS;
	}

	hipStream_t hip_stream;
	CUTF_CHECK_ERROR(hipblasGetStream(handle, &hip_stream));

	mtk::ozimmu::CULiP::profile_result profile_result;
	const auto profiling_flag = mtk::ozimmu::CULiP::is_profiling_enabled();

	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, const void*, hipDataType, int, const void*, void*, hipDataType, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = ozIMMU_get_function_pointer(
			cublas_library_name.c_str(),
			__func__
			);

	if (profiling_flag) {
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%d-n%d-k%d", __func__, mtk::ozimmu::CULiP::get_cublasOperation_t_string(transa), mtk::ozimmu::CULiP::get_cublasOperation_t_string(transb), m, n, k);
		mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
	}

	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, B, Btype, ldb, beta, C, Ctype, ldc, computeType, algo);

	if (profiling_flag) {
		// Record end rimestamp
		mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::print_profile_result, (void*)&profile_result);
	}

	return res;
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDgemm (hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const double *alpha,
		const double *A,
		int lda,
		const double *B,
		int ldb,
		const double *beta,
		double *C,
		int ldc) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	hipblasMath_t math_mode;
	CUTF_CHECK_ERROR(hipblasGetMathMode(handle, &math_mode));
	return hipblasGemmEx(
			handle,
			transa, transb,
			m, n, k,
			alpha,
			A, HIP_R_64F, lda,
			B, HIP_R_64F, ldb,
			beta,
			C, HIP_R_64F, ldc,
			HIPBLAS_COMPUTE_64F,
			(math_mode == HIPBLAS_DEFAULT_MATH ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP)
			);
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZgemm (hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const hipDoubleComplex *alpha,
		const hipDoubleComplex *A,
		int lda,
		const hipDoubleComplex *B,
		int ldb,
		const hipDoubleComplex *beta,
		hipDoubleComplex *C,
		int ldc) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	hipblasMath_t math_mode;
	CUTF_CHECK_ERROR(hipblasGetMathMode(handle, &math_mode));
	return hipblasGemmEx(
			handle,
			transa, transb,
			m, n, k,
			alpha,
			A, HIP_C_64F, lda,
			B, HIP_C_64F, ldb,
			beta,
			C, HIP_C_64F, ldc,
			HIPBLAS_COMPUTE_64F,
			(math_mode == HIPBLAS_DEFAULT_MATH ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP)
			);
#endif
}

CUBLASAPI hipblasStatus_t hipblasGemmStridedBatchedEx(hipblasHandle_t handle, hipblasOperation_t transa,
		hipblasOperation_t transb, int m, int n, int k,
		const void *alpha, const void *A,
		hipDataType Atype, int lda, long long int strideA, const void *B,
		hipDataType Btype, int ldb, long long int strideB, const void *beta,
		void *C, hipDataType Ctype, int ldc, long long int strideC,
		int batch_count,
		hipblasComputeType_t computeType,
		hipblasGemmAlgo_t algo) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	const auto compute_mode = get_compute_mode(m, n, k);
	if (compute_mode != mtk::ozimmu::dgemm && m >= 1024 && n >= 1024 && k >= 1024 &&
			((Atype == HIP_R_64F && Btype == HIP_R_64F && Ctype == HIP_R_64F) || (Atype == HIP_C_64F && Btype == HIP_C_64F && Ctype == HIP_C_64F))
			) {
		const auto gemm_config = mtk::ozimmu::gemm_list_t {
			std::tuple<std::size_t, std::size_t, std::size_t, mtk::ozimmu::element_kind_t, mtk::ozimmu::compute_mode_t>{m, n, k, mtk::ozimmu::complx, compute_mode}
		};

		hipStream_t hip_stream;
		CUTF_CHECK_ERROR(hipblasGetStream(handle, &hip_stream));
		mtk::ozimmu::set_cuda_stream(get_global_ozimmu_handle(), hip_stream);

		mtk::ozimmu::CULiP::profile_result profile_result;
		const auto profiling_flag = mtk::ozimmu::CULiP::is_profiling_enabled();

		if (profiling_flag) {
			snprintf(profile_result.function_name, profile_result.function_name_length - 1,
							 "%s%s_stridedBatched-%s%s-m%d-n%d-k%d-batch_count%d",
							 (Atype == HIP_R_64F ? "D" : "Z"),
							 mtk::ozimmu::get_compute_mode_name_str(compute_mode).c_str(),
							 mtk::ozimmu::CULiP::get_cublasOperation_t_string(transa), mtk::ozimmu::CULiP::get_cublasOperation_t_string(transb), m, n, k, batch_count);
			mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
		}

		int error = 0;
		for (int batch_id = 0; batch_id < batch_count; batch_id++) {
			if (Atype == HIP_R_64F) {
				error = mtk::ozimmu::gemm(
						get_global_ozimmu_handle(),
						op_cublas2oz(transa),
						op_cublas2oz(transb),
						m, n, k,
						reinterpret_cast<const double*>(alpha),
						reinterpret_cast<const double*>(A) + strideA * batch_id, lda,
						reinterpret_cast<const double*>(B) + strideB * batch_id, ldb,
						reinterpret_cast<const double*>(beta),
						reinterpret_cast<double*>(C) + strideC * batch_id, ldc,
						compute_mode,
						mtk::ozimmu::real
						);
			} else if (Atype == HIP_C_64F) {
				error = mtk::ozimmu::gemm(
						get_global_ozimmu_handle(),
						op_cublas2oz(transa),
						op_cublas2oz(transb),
						m, n, k,
						reinterpret_cast<const hipDoubleComplex*>(alpha),
						reinterpret_cast<const hipDoubleComplex*>(A) + strideA * batch_id, lda,
						reinterpret_cast<const hipDoubleComplex*>(B) + strideB * batch_id, ldb,
						reinterpret_cast<const hipDoubleComplex*>(beta),
						reinterpret_cast<hipDoubleComplex*>(C) + strideC * batch_id, ldc,
						compute_mode,
						mtk::ozimmu::complx
						);
			}
			if (error) {
				break;
			}
		}

		if (profiling_flag) {
			// Record end rimestamp
			mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

			// Print result
			mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::print_profile_result, (void*)&profile_result);
		}

		if (error) {
			return HIPBLAS_STATUS_INTERNAL_ERROR;
		}

		return HIPBLAS_STATUS_SUCCESS;
	}

	hipStream_t hip_stream;
	CUTF_CHECK_ERROR(hipblasGetStream(handle, &hip_stream));

	mtk::ozimmu::CULiP::profile_result profile_result;
	const auto profiling_flag = mtk::ozimmu::CULiP::is_profiling_enabled();

	hipblasStatus_t (*func_ptr)(hipblasHandle_t, hipblasOperation_t, hipblasOperation_t, int, int, int, const void*, const void*, hipDataType, int, long long int, const void*, hipDataType, int, long long int, const void*, void*, hipDataType, int, long long int, int, hipblasComputeType_t, hipblasGemmAlgo_t);
	*(void**)(&func_ptr) = ozIMMU_get_function_pointer(
			cublas_library_name.c_str(),
			__func__
			);

	if (profiling_flag) {
		snprintf(profile_result.function_name, profile_result.function_name_length - 1, "%s-%s%s-m%d-n%d-k%d", __func__, mtk::ozimmu::CULiP::get_cublasOperation_t_string(transa), mtk::ozimmu::CULiP::get_cublasOperation_t_string(transb), m, n, k);
		mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::record_timestamp, (void*)&profile_result.start_timestamp);
	}

	const auto res = (*func_ptr)(handle, transa, transb, m, n, k, alpha, A, Atype, lda, strideA, B, Btype, ldb, strideB, beta, C, Ctype, ldc, strideC, batch_count, computeType, algo);

	if (profiling_flag) {
		// Record end rimestamp
		mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::record_timestamp, (void*)&profile_result.end_timestamp);

		// Print result
		mtk::ozimmu::CULiP::launch_function(hip_stream, &mtk::ozimmu::CULiP::print_profile_result, (void*)&profile_result);
	}

	return res;
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasDgemmStridedBatched(hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const double *alpha,
		const double *A,
		int lda,
		long long int strideA,
		const double *B,
		int ldb,
		long long int strideB,
		const double *beta,
		double *C,
		int ldc,
		long long int strideC,
		int batch_count
		) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	hipblasMath_t math_mode;
	CUTF_CHECK_ERROR(hipblasGetMathMode(handle, &math_mode));
	return hipblasGemmStridedBatchedEx(
			handle,
			transa, transb,
			m, n, k,
			alpha,
			A, HIP_R_64F, lda, strideA,
			B, HIP_R_64F, ldb, strideB,
			beta,
			C, HIP_R_64F, ldc, strideC,
			batch_count,
			HIPBLAS_COMPUTE_64F,
			(math_mode == HIPBLAS_DEFAULT_MATH ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP)
			);
#endif
}

CUBLASAPI hipblasStatus_t CUBLASWINAPI hipblasZgemmStridedBatched(hipblasHandle_t handle,
		hipblasOperation_t transa,
		hipblasOperation_t transb,
		int m,
		int n,
		int k,
		const hipDoubleComplex *alpha,
		const hipDoubleComplex *A,
		int lda,
		long long int strideA,
		const hipDoubleComplex *B,
		int ldb,
		long long int strideB,
		const hipDoubleComplex *beta,
		hipDoubleComplex *C,
		int ldc,
		long long int strideC,
		int batch_count
		) {
#ifdef __CUDA_ARCH__
	return HIPBLAS_STATUS_NOT_SUPPORTED;
#else
	hipblasMath_t math_mode;
	CUTF_CHECK_ERROR(hipblasGetMathMode(handle, &math_mode));
	return hipblasGemmStridedBatchedEx(
			handle,
			transa, transb,
			m, n, k,
			alpha,
			A, HIP_C_64F, lda, strideA,
			B, HIP_C_64F, ldb, strideB,
			beta,
			C, HIP_C_64F, ldc, strideC,
			batch_count,
			HIPBLAS_COMPUTE_64F,
			(math_mode == HIPBLAS_DEFAULT_MATH ? HIPBLAS_GEMM_DEFAULT : CUBLAS_GEMM_DEFAULT_TENSOR_OP)
			);
#endif
}
} // extern "C"
