#include "hip/hip_runtime.h"
#include <algorithm>
#include <iostream>
#include <chrono>
#include <oztcecgemm/oztcecgemm.hpp>
#include <cutf/memory.hpp>
#include <cutf/hiprand.hpp>
#include <cutf/curand_kernel.hpp>
#include <cutf/device.hpp>
#include <cutf/math.hpp>
#include <mateval/comparison_cuda.hpp>
#include <matfile/matfile.hpp>
#include <gpu_monitor/gpu_monitor.hpp>

constexpr unsigned test_count = 100;

constexpr unsigned long long seed = 0;

std::string get_gpu_name_str() {
	const auto device_prop = cutf::device::get_properties_vector()[0];
	std::string gpu_name = device_prop.name;
	std::replace(gpu_name.begin(), gpu_name.end(), ' ', '_');
	return gpu_name;
}

inline mtk::mateval::layout_t conv_layout_oz2mateval(const mtk::oztcecgemm::operation_t op) {
	if (op == mtk::oztcecgemm::op_n) {
		return mtk::mateval::col_major;
	}
	return mtk::mateval::row_major;
}

template <class T>
__global__ void adjust_urand_kernel(
		T* const ptr,
		const T min_urand,
		const T max_urand,
		const std::size_t n
		) {
	const auto tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) {
		return;
	}

	const auto v = ptr[tid];
	ptr[tid] = v * (max_urand - min_urand) + min_urand;
}

template <class T>
void adjust_urand(
		T* const ptr,
		const T min_urand,
		const T max_urand,
		const std::size_t n
		) {
	const auto block_size = 256lu;
	const auto grid_size = (n + block_size - 1) / block_size;

	adjust_urand_kernel<T><<<grid_size, block_size>>>(
			ptr,
			min_urand, max_urand,
			n
			);
}

// See "5.1 Experimental Settings" of https://link.springer.com/chapter/10.1007/978-3-030-50743-5_12
template <class T>
__global__ void gen_exp_rand_kernel(
		T* const ptr,
		const std::size_t N,
		const T phi,
		const std::uint64_t seed
		) {
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandStateXORWOW_t curand_state;
	hiprand_init(seed, tid, 0, &curand_state);

	for (std::size_t i = tid; i < N; i += blockDim.x * gridDim.x){
		const auto rand  = cutf::curand_kernel::uniform<T>(&curand_state);
		const auto randn = cutf::curand_kernel::normal<T>(&curand_state);
		const auto v = (rand - static_cast<T>(0.5)) * exp(phi * randn);

		ptr[i] = v;
	}
}

template <class T>
void gen_exp_rand(
		T* const ptr,
		const std::size_t N,
		const T phi,
		const std::uint64_t seed
		) {
	const std::size_t block_size = 256;
	const std::size_t grid_size = std::min((N + block_size - 1) / block_size, 1024lu);

	gen_exp_rand_kernel<<<grid_size, block_size>>>(
			ptr,
			N,
			phi,
			seed
			);
}

template <class C_T, class AB_T, class MATMUL_FUNC>
void gemm_eval_core(
		const mtk::oztcecgemm::operation_t op_a,
		const mtk::oztcecgemm::operation_t op_b,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const AB_T* const a_ptr, const std::size_t lda,
		const AB_T* const b_ptr, const std::size_t ldb,
		C_T* const c_ptr, const std::size_t ldc,
		const MATMUL_FUNC matmul_func,
		const mtk::oztcecgemm::compute_mode_t mode,
		const std::string input_mode
		) {
	matmul_func(
			op_a, op_b,
			m, n, k,
			a_ptr, lda,
			b_ptr, ldb,
			c_ptr, ldc
			);

	mtk::mateval::error_map_t error;
	if (mtk::oztcecgemm::get_output_type(mode) == mtk::oztcecgemm::fp32) {
		error = mtk::mateval::cuda::get_error_AxB(
				mtk::mateval::relative_residual | mtk::mateval::max_relative_error,
				m, n, k,
				conv_layout_oz2mateval(op_a),
				conv_layout_oz2mateval(op_b),
				mtk::mateval::col_major,
				a_ptr, lda,
				b_ptr, ldb,
				reinterpret_cast<float*>(c_ptr), ldc
				);
	} else {
		error = mtk::mateval::cuda::get_error_AxB(
				mtk::mateval::relative_residual | mtk::mateval::max_relative_error,
				m, n, k,
				conv_layout_oz2mateval(op_a),
				conv_layout_oz2mateval(op_b),
				mtk::mateval::col_major,
				a_ptr, lda,
				b_ptr, ldb,
				reinterpret_cast<double*>(c_ptr), ldc
				);
	}

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto start_clock = std::chrono::system_clock::now();

	for (unsigned i = 0; i < test_count; i++) {
		matmul_func(
				op_a, op_b,
				m, n, k,
				a_ptr, lda,
				b_ptr, ldb,
				c_ptr, ldc
				);
	}

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();

	const auto elapsed_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_clock - start_clock).count() * 1e-9 / test_count;

	const auto throughput = 2 * m * n * k / elapsed_time;

	std::printf("%s,%s,%s,%lu,%lu,%lu,%e,%e,%e\n",
			get_gpu_name_str().c_str(),
			input_mode.c_str(),
			mtk::oztcecgemm::get_compute_mode_name_str(mode).c_str(),
			m, n, k,
			error.at(mtk::mateval::relative_residual),
			error.at(mtk::mateval::max_relative_error),
			throughput * 1e-12
			);
	std::fflush(stdout);
}

template <class T>
void gemm_eval(
		const mtk::oztcecgemm::gemm_list_t& gemm_list,
		const std::string input_mode
		) {
	mtk::oztcecgemm::handle_t oztcecgemm_handle;
	mtk::oztcecgemm::create(&oztcecgemm_handle);
	mtk::oztcecgemm::reallocate_working_memory(oztcecgemm_handle, gemm_list);

	std::size_t max_AB_count = 0;
	std::size_t max_C_size = 0;
	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		max_AB_count = std::max(max_AB_count, m * k + k * n);
		max_C_size  = std::max(max_C_size , m * n *
				mtk::oztcecgemm::get_data_size_in_byte(
				mtk::oztcecgemm::get_output_type(std::get<3>(gemm))));
	}

	auto mat_AB_uptr = cutf::memory::get_device_unique_ptr<T>(max_AB_count);
	auto mat_C_uptr  = cutf::memory::get_device_unique_ptr<std::uint8_t>(max_C_size);

	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_MT19937);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));
	if (input_mode == "normal01") {
		CUTF_CHECK_ERROR(cutf::hiprand::generate_normal(*cugen.get(), mat_AB_uptr.get(), max_AB_count, 0, 1));
	} else if (input_mode == "urand01") {
		CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), mat_AB_uptr.get(), max_AB_count));
	} else {
		double phi = 0;
		try {
			phi = std::stod(input_mode.substr(9));
		} catch (const std::exception& e) {
			std::fprintf(stderr, "Error: %s [%s (line:%d)]\n", e.what(), __FILE__, __LINE__);
			return;
		}
		gen_exp_rand<T>(mat_AB_uptr.get(), max_AB_count, phi, 0);
	}

	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		const auto mode = std::get<3>(gemm);
		gemm_eval_core(
				mtk::oztcecgemm::op_n,
				mtk::oztcecgemm::op_n,
				m, n, k,
				mat_AB_uptr.get(), m,
				mat_AB_uptr.get() + m * k, k,
				mat_C_uptr.get(), m,
				[&](
						const mtk::oztcecgemm::operation_t op_a,
						const mtk::oztcecgemm::operation_t op_b,
						const std::size_t m,
						const std::size_t n,
						const std::size_t k,
						const T* const a_ptr, const std::size_t lda,
						const T* const b_ptr, const std::size_t ldb,
						void* const c_ptr, const std::size_t ldc
									) {
					if (mtk::oztcecgemm::get_output_type(mode) == mtk::oztcecgemm::fp32) {
						using C_T = float;
						const C_T alpha = 1, beta = 0;
						mtk::oztcecgemm::gemm(
								oztcecgemm_handle,
								op_a, op_b,
								m, n, k,
								&alpha,
								a_ptr, lda,
								b_ptr, ldb,
								&beta,
								c_ptr, ldc,
								mode
								);
					} else {
						using C_T = double;
						const C_T alpha = 1, beta = 0;
						mtk::oztcecgemm::gemm(
								oztcecgemm_handle,
								op_a, op_b,
								m, n, k,
								&alpha,
								a_ptr, lda,
								b_ptr, ldb,
								&beta,
								c_ptr, ldc,
								mode
								);
					}
				},
				mode,
				input_mode
				);
	}

	mtk::oztcecgemm::destroy(oztcecgemm_handle);
}

template <class SRC_T, class DST_T>
__global__ void vector_copy_kernel(
		DST_T* const dst_ptr,
		const SRC_T* const src_ptr,
		const std::size_t N
		) {
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= N) {
		return;
	}

	dst_ptr[tid] = src_ptr[tid];
}

template <class DEVICE_T>
void matfile_to_device_memory(
		DEVICE_T* const d_ptr,
		const std::string matfile_path
		) {
	std::size_t m, n;
	mtk::matfile::load_size(m, n, matfile_path);
	const auto dtype = mtk::matfile::load_dtype(matfile_path);

	auto h_mat_uptr = cutf::memory::get_host_unique_ptr<std::uint8_t>(m * n * mtk::matfile::get_dtype_size(dtype));

	const std::size_t block_size = 256;
	const std::size_t grid_size = (m * n + block_size - 1) / block_size;

	if (dtype == mtk::matfile::fp32) {
		mtk::matfile::load_dense(
				reinterpret_cast<float*>(h_mat_uptr.get()),
				m,
				matfile_path
				);
		vector_copy_kernel<<<grid_size, block_size>>>(
				d_ptr,
				reinterpret_cast<float*>(h_mat_uptr.get()),
				m * n
				);
	} else if (dtype == mtk::matfile::fp64) {
		mtk::matfile::load_dense(
				reinterpret_cast<double*>(h_mat_uptr.get()),
				m,
				matfile_path
				);
		vector_copy_kernel<<<grid_size, block_size>>>(
				d_ptr,
				reinterpret_cast<double*>(h_mat_uptr.get()),
				m * n
				);
	} else {
		mtk::matfile::load_dense(
				reinterpret_cast<long double*>(h_mat_uptr.get()),
				m,
				matfile_path
				);
		vector_copy_kernel<<<grid_size, block_size>>>(
				d_ptr,
				reinterpret_cast<long double*>(h_mat_uptr.get()),
				m * n
				);
	}
	CUTF_CHECK_ERROR(hipDeviceSynchronize());
}

template <class C_T>
mtk::mateval::error_map_t eval_matfile(
		const std::string matfile_C_path,
		const C_T* const c_ptr
		) {
	std::size_t m, n;
	mtk::matfile::load_size(m, n, matfile_C_path);
	const auto dtype = mtk::matfile::load_dtype(matfile_C_path);

	mtk::mateval::error_map_t error;
	if (dtype == mtk::matfile::fp32) {
		using R_T = float;
		auto mat_ref_uptr = cutf::memory::get_host_unique_ptr<R_T>(m * n);
		mtk::matfile::load_dense(mat_ref_uptr.get(), m, matfile_C_path);

		error = mtk::mateval::cuda::get_error(
				mtk::mateval::max_relative_error | mtk::mateval::relative_residual,
				m, n,
				mtk::mateval::col_major,
				mtk::mateval::col_major,
				c_ptr, m,
				reinterpret_cast<R_T*>(mat_ref_uptr.get()), m
				);
	} else if (dtype == mtk::matfile::fp64) {
		using R_T = double;
		auto mat_ref_uptr = cutf::memory::get_host_unique_ptr<R_T>(m * n);
		mtk::matfile::load_dense(mat_ref_uptr.get(), m, matfile_C_path);

		error = mtk::mateval::cuda::get_error(
				mtk::mateval::max_relative_error | mtk::mateval::relative_residual,
				m, n,
				mtk::mateval::col_major,
				mtk::mateval::col_major,
				c_ptr, m,
				reinterpret_cast<R_T*>(mat_ref_uptr.get()), m
				);
	}

	return error;
}

template <class T>
void gemm_eval_matfile(
		const mtk::oztcecgemm::gemm_list_t& gemm_list,
		const std::string matfile_A_path,
		const std::string matfile_B_path
		) {
	mtk::oztcecgemm::handle_t oztcecgemm_handle;
	mtk::oztcecgemm::create(&oztcecgemm_handle);
	mtk::oztcecgemm::reallocate_working_memory(oztcecgemm_handle, gemm_list);

	std::size_t max_AB_count = 0;
	std::size_t max_C_size = 0;
	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		max_AB_count = std::max(max_AB_count, m * k + k * n);
		max_C_size  = std::max(max_C_size , m * n *
				mtk::oztcecgemm::get_data_size_in_byte(
				mtk::oztcecgemm::get_output_type(std::get<3>(gemm))));
	}

	auto mat_AB_uptr = cutf::memory::get_device_unique_ptr<T>(max_AB_count);
	auto mat_C_uptr  = cutf::memory::get_device_unique_ptr<std::uint8_t>(max_C_size);


	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		const auto mode = std::get<3>(gemm);

		const auto a_ptr = mat_AB_uptr.get();
		const auto b_ptr = mat_AB_uptr.get() + m * k;
		const auto c_ptr = mat_C_uptr.get();

		matfile_to_device_memory(a_ptr, matfile_A_path);
		matfile_to_device_memory(b_ptr, matfile_B_path);

		gemm_eval_core(
				mtk::oztcecgemm::op_n,
				mtk::oztcecgemm::op_n,
				m, n, k,
				mat_AB_uptr.get(), m,
				mat_AB_uptr.get() + m * k, k,
				mat_C_uptr.get(), m,
				[&](
						const mtk::oztcecgemm::operation_t op_a,
						const mtk::oztcecgemm::operation_t op_b,
						const std::size_t m,
						const std::size_t n,
						const std::size_t k,
						const T* const a_ptr, const std::size_t lda,
						const T* const b_ptr, const std::size_t ldb,
						void* const c_ptr, const std::size_t ldc
									) {
					if (mtk::oztcecgemm::get_output_type(mode) == mtk::oztcecgemm::fp32) {
						using C_T = float;
						const C_T alpha = 1, beta = 0;
						mtk::oztcecgemm::gemm(
								oztcecgemm_handle,
								op_a, op_b,
								m, n, k,
								&alpha,
								a_ptr, lda,
								b_ptr, ldb,
								&beta,
								c_ptr, ldc,
								mode
								);
					} else {
						using C_T = double;
						const C_T alpha = 1, beta = 0;
						mtk::oztcecgemm::gemm(
								oztcecgemm_handle,
								op_a, op_b,
								m, n, k,
								&alpha,
								a_ptr, lda,
								b_ptr, ldb,
								&beta,
								c_ptr, ldc,
								mode
								);
					}
				},
				mode,
				"matfile"
				);
	}

	mtk::oztcecgemm::destroy(oztcecgemm_handle);
}

template <class T>
void gemm_eval_power(
		const mtk::oztcecgemm::gemm_list_t& gemm_list
		) {
	mtk::oztcecgemm::handle_t oztcecgemm_handle;
	mtk::oztcecgemm::create(&oztcecgemm_handle);
	mtk::oztcecgemm::reallocate_working_memory(oztcecgemm_handle, gemm_list);

	std::size_t max_AB_count = 0;
	std::size_t max_C_size = 0;
	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		max_AB_count = std::max(max_AB_count, m * k + k * n);
		max_C_size  = std::max(max_C_size , m * n *
				mtk::oztcecgemm::get_data_size_in_byte(
				mtk::oztcecgemm::get_output_type(std::get<3>(gemm))));
	}

	auto mat_AB_uptr = cutf::memory::get_device_unique_ptr<T>(max_AB_count);
	auto mat_C_uptr  = cutf::memory::get_device_unique_ptr<std::uint8_t>(max_C_size);

	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_MT19937);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));

	CUTF_CHECK_ERROR(cutf::hiprand::generate_normal(*cugen.get(), mat_AB_uptr.get(), max_AB_count, 0, 1));

	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		const auto mode = std::get<3>(gemm);

		const auto gemm_func = [&](
				const mtk::oztcecgemm::operation_t op_a,
				const mtk::oztcecgemm::operation_t op_b,
				const std::size_t m,
				const std::size_t n,
				const std::size_t k,
				const T* const a_ptr, const std::size_t lda,
				const T* const b_ptr, const std::size_t ldb,
				void* const c_ptr, const std::size_t ldc
				) {
			if (mtk::oztcecgemm::get_output_type(mode) == mtk::oztcecgemm::fp32) {
				using C_T = float;
				const C_T alpha = 1, beta = 0;
				mtk::oztcecgemm::gemm(
						oztcecgemm_handle,
						op_a, op_b,
						m, n, k,
						&alpha,
						a_ptr, lda,
						b_ptr, ldb,
						&beta,
						c_ptr, ldc,
						mode
						);
			} else {
				using C_T = double;
				const C_T alpha = 1, beta = 0;
				mtk::oztcecgemm::gemm(
						oztcecgemm_handle,
						op_a, op_b,
						m, n, k,
						&alpha,
						a_ptr, lda,
						b_ptr, ldb,
						&beta,
						c_ptr, ldc,
						mode
						);
			}
		};

		constexpr std::size_t duration_time = 10;
		std::size_t c = 0;
		const auto result = mtk::gpu_monitor::measure_power_consumption(
				[&]() {
					CUTF_CHECK_ERROR(hipDeviceSynchronize());
					const auto start_clock = std::chrono::system_clock::now();
					while (true) {
						gemm_func(
								mtk::oztcecgemm::op_n,
								mtk::oztcecgemm::op_n,
								m, n, k,
								mat_AB_uptr.get(), m,
								mat_AB_uptr.get() + m * k, k,
								mat_C_uptr.get(), m
								);
						if (((++c) % 10) == 0) {
							CUTF_CHECK_ERROR(hipDeviceSynchronize());
							const auto current_clock = std::chrono::system_clock::now();
							const auto elapsed_time = std::chrono::duration_cast<std::chrono::microseconds>(current_clock - start_clock).count() * 1e-6;
							if (elapsed_time > duration_time) {
								break;
							}
						}
					}
				},
				100
				);
		const auto power = mtk::gpu_monitor::get_integrated_power_consumption(result);
		const auto elapsed_time = mtk::gpu_monitor::get_elapsed_time(result);
		const auto average_power = power / elapsed_time;
		const auto flops_per_watt = 2lu * m * n * k * c / power;
		const auto throughput_in_tflops = 2lu * m * n * k * c / elapsed_time * 1e-12;

		std::printf("%s,%s,%lu,%lu,%lu,%e,%e,%e,%e,%lu\n",
				get_gpu_name_str().c_str(),
				mtk::oztcecgemm::get_compute_mode_name_str(mode).c_str(),
				m, n, k,
				throughput_in_tflops,
				average_power,
				flops_per_watt * 1e-9,
				elapsed_time,
				c
				);
		std::fflush(stdout);
	}

	mtk::oztcecgemm::destroy(oztcecgemm_handle);
}

std::vector<mtk::oztcecgemm::compute_mode_t> get_supported_compute_mode() {
	return std::vector<mtk::oztcecgemm::compute_mode_t>{
		mtk::oztcecgemm::sgemm,
		mtk::oztcecgemm::dgemm,
		mtk::oztcecgemm::fp64_int8_6,
		mtk::oztcecgemm::fp64_int8_7,
		mtk::oztcecgemm::fp64_int8_8,
		mtk::oztcecgemm::fp64_int8_9,
		mtk::oztcecgemm::fp64_int8_10,
		mtk::oztcecgemm::fp64_int8_11,
		mtk::oztcecgemm::fp64_int8_12,
		mtk::oztcecgemm::fp64_int8_13,
	};
}

std::vector<mtk::oztcecgemm::compute_mode_t> get_compute_mode_list_from_argv(
		const std::size_t count,
		char** argv
		) {
	std::vector<mtk::oztcecgemm::compute_mode_t> mode_list;

	for (std::size_t i = 0; i < count; i++) {
		bool added = false;
		for (const auto m : get_supported_compute_mode()) {
			if (std::string(argv[i]) == mtk::oztcecgemm::get_compute_mode_name_str(m)) {
				mode_list.push_back(m);
				added = true;
				break;
			}
		}
		if (!added) {
			std::fprintf(stderr, "Warning: Unknown compute mode \"%s\"\n", argv[i]);
		}
	}

	return mode_list;
}

void print_usage(
		const char* const program_name
		) {
	std::string compute_mode_list_str = "";
	for (const auto& name : get_supported_compute_mode()) {
		compute_mode_list_str += mtk::oztcecgemm::get_compute_mode_name_str(name) + " ";
	}

	std::printf(
			"Usage:\n"
			"%s matfile [/path/to/A.matrix] [/path/to/B.matrix] [Computing mode list]\n"
			"%s [urand01 | normal01 | exp_rand-X] [seq|exp2] [start_N] [end_N] [interval_N] [Computing mode list]\n"
			"%s power [seq|exp2] [start_N] [end_N] [interval_N] [Computing mode list]\n"
			"Compute modes:\n"
			" %s\n",
			program_name,
			program_name,
			program_name,
			compute_mode_list_str.c_str()
			);
}

int main(int argc, char** argv) {

	if (argc <= 2) {
		print_usage(argv[0]);
		return 1;
	}

	const auto input_mode = std::string(argv[1]);
	if (input_mode == "matfile") {
		if (argc <= 4) {
			print_usage(argv[0]);
			return 1;
		}

		const auto matfile_A_path = std::string(argv[2]);
		const auto matfile_B_path = std::string(argv[3]);
		const auto compute_mode_list = get_compute_mode_list_from_argv(argc - 4, argv + 4);

		std::size_t am, an, bm, bn;
		mtk::matfile::load_size(am, an, matfile_A_path);
		mtk::matfile::load_size(bm, bn, matfile_B_path);
		if (an != bm) {
			std::fprintf(stderr, "Error: matrix shapes are mismatch: A=(%lu, %lu), B=(%lu, %lu), C=(%lu, %lu)\n",
					am, an,
					bm, bn,
					am, bn
					);
			return 1;
		}

		mtk::oztcecgemm::gemm_list_t fp32in_gemm_list;
		mtk::oztcecgemm::gemm_list_t fp64in_gemm_list;

		for (auto compute_mode : compute_mode_list) {
			if (mtk::oztcecgemm::get_output_type(compute_mode) == mtk::oztcecgemm::fp32) {
				fp32in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
							am,
							bn,
							an,
							compute_mode
							));
			} else {
				fp64in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
							am,
							bn,
							an,
							compute_mode
							));
			}
		}

		std::printf(
				"matfile test:\n"
				"A : %s\n"
				"B : %s\n",
				matfile_A_path.c_str(),
				matfile_B_path.c_str()
				);
		std::printf("gpu,input,mode,m,n,k,residual,max_relative\n");
		std::fflush(stdout);
		if (fp32in_gemm_list.size() != 0) {
			gemm_eval_matfile<float>(fp32in_gemm_list, matfile_A_path, matfile_B_path);
		}
		if (fp64in_gemm_list.size() != 0) {
			gemm_eval_matfile<double>(fp64in_gemm_list, matfile_A_path, matfile_B_path);
		}
	} else if (input_mode == "urand01" || input_mode == "normal01" || (input_mode.length() >= 9 && input_mode.substr(0, 9) == "exp_rand-")) {
		if (argc <= 6) {
			print_usage(argv[0]);
			return 1;
		}
		const auto N_mode = std::string(argv[2]);
		if (N_mode != "seq" && N_mode != "exp2") {
			std::fprintf(stderr, "Error: unknown N mode \"%s\"\n", N_mode.c_str());
			return 1;
		}
		const auto min_N = std::stoul(argv[3]);
		const auto max_N = std::stoul(argv[4]);
		const auto interval_N = std::stoul(argv[5]);
		const auto compute_mode_list = get_compute_mode_list_from_argv(argc - 6, argv + 6);

		mtk::oztcecgemm::gemm_list_t fp32in_gemm_list;
		mtk::oztcecgemm::gemm_list_t fp64in_gemm_list;

		for (std::size_t N = min_N; N <= max_N; N += interval_N) {
			auto real_N = N;
			if (N_mode == "exp2") {real_N = 1lu << N;}

			for (auto compute_mode : compute_mode_list) {
				if (mtk::oztcecgemm::get_output_type(compute_mode) == mtk::oztcecgemm::fp32) {
					fp32in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
								real_N,
								real_N,
								real_N,
								compute_mode
								));
				} else {
					fp64in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
								real_N,
								real_N,
								real_N,
								compute_mode
								));
				}
			}
		}

		std::printf("gpu,input,mode,m,n,k,residual,max_relative,throughput_in_tflops\n");
		std::fflush(stdout);
		if (fp32in_gemm_list.size() != 0) {
			gemm_eval<float>(fp32in_gemm_list, input_mode);
		}
		if (fp64in_gemm_list.size() != 0) {
			gemm_eval<double>(fp64in_gemm_list, input_mode);
		}
	} else if (input_mode == "power") {
		if (argc <= 6) {
			print_usage(argv[0]);
			return 1;
		}
		const auto N_mode = std::string(argv[2]);
		if (N_mode != "seq" && N_mode != "exp2") {
			std::fprintf(stderr, "Error: unknown N mode \"%s\"\n", N_mode.c_str());
			return 1;
		}
		const auto min_N = std::stoul(argv[3]);
		const auto max_N = std::stoul(argv[4]);
		const auto interval_N = std::stoul(argv[5]);
		const auto compute_mode_list = get_compute_mode_list_from_argv(argc - 6, argv + 6);

		mtk::oztcecgemm::gemm_list_t fp32in_gemm_list;
		mtk::oztcecgemm::gemm_list_t fp64in_gemm_list;

		for (std::size_t N = min_N; N <= max_N; N += interval_N) {
			auto real_N = N;
			if (N_mode == "exp2") {real_N = 1lu << N;}

			for (auto compute_mode : compute_mode_list) {
				if (mtk::oztcecgemm::get_output_type(compute_mode) == mtk::oztcecgemm::fp32) {
					fp32in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
								real_N,
								real_N,
								real_N,
								compute_mode
								));
				} else {
					fp64in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
								real_N,
								real_N,
								real_N,
								compute_mode
								));
				}
			}
		}

		std::printf("gpu,mode,m,n,k,throughput_in_tflops,avg_watt,gflops_per_watt,time,count\n");
		std::fflush(stdout);
		if (fp32in_gemm_list.size() != 0) {
			gemm_eval_power<float>(fp32in_gemm_list);
		}
		if (fp64in_gemm_list.size() != 0) {
			gemm_eval_power<double>(fp64in_gemm_list);
		}
	} else {
		std::fprintf(stderr, "Error: Unknown input mode \"%s\"\n", input_mode.c_str());
		return 1;
	}
}
