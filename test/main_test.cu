#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <oztcecgemm/oztcecgemm.hpp>
#include <cutf/memory.hpp>
#include <cutf/hiprand.hpp>
#include <mateval/comparison_cuda.hpp>

constexpr unsigned test_count = 100;

inline mtk::mateval::layout_t conv_layout_oz2mateval(const mtk::oztcecgemm::operation_t op) {
	if (op == mtk::oztcecgemm::op_n) {
		return mtk::mateval::col_major;
	}
	return mtk::mateval::row_major;
}

template <class C_T, class AB_T, class MATMUL_FUNC>
void gemm_eval_core(
		const mtk::oztcecgemm::operation_t op_a,
		const mtk::oztcecgemm::operation_t op_b,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const AB_T* const a_ptr, const std::size_t lda,
		const AB_T* const b_ptr, const std::size_t ldb,
		C_T* const c_ptr, const std::size_t ldc,
		const MATMUL_FUNC matmul_func,
		const mtk::oztcecgemm::compute_mode_t mode
		) {
	matmul_func(
			op_a, op_b,
			m, n, k,
			a_ptr, lda,
			b_ptr, ldb,
			c_ptr, ldc
			);

	mtk::mateval::error_map_t error;
	if (mtk::oztcecgemm::get_output_type(mode) == mtk::oztcecgemm::fp32) {
		error = mtk::mateval::cuda::get_error_AxB(
				mtk::mateval::relative_residual | mtk::mateval::max_relative_error,
				m, n, k,
				conv_layout_oz2mateval(op_a),
				conv_layout_oz2mateval(op_b),
				mtk::mateval::col_major,
				a_ptr, lda,
				b_ptr, ldb,
				reinterpret_cast<float*>(c_ptr), ldc
				);
	} else {
		error = mtk::mateval::cuda::get_error_AxB(
				mtk::mateval::relative_residual | mtk::mateval::max_relative_error,
				m, n, k,
				conv_layout_oz2mateval(op_a),
				conv_layout_oz2mateval(op_b),
				mtk::mateval::col_major,
				a_ptr, lda,
				b_ptr, ldb,
				reinterpret_cast<double*>(c_ptr), ldc
				);
	}

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto start_clock = std::chrono::system_clock::now();

	for (unsigned i = 0; i < test_count; i++) {
		matmul_func(
				op_a, op_b,
				m, n, k,
				a_ptr, lda,
				b_ptr, ldb,
				c_ptr, ldc
				);
	}

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();

	const auto elapsed_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_clock - start_clock).count() * 1e-9 / test_count;

	const auto throughput = 2 * m * n * k / elapsed_time;

	std::printf("%s,%lu,%lu,%lu,%e,%e,%e\n",
			mtk::oztcecgemm::get_compute_mode_name_str(mode).c_str(),
			m, n, k,
			error.at(mtk::mateval::relative_residual),
			error.at(mtk::mateval::max_relative_error),
			throughput * 1e-12
			);
	std::fflush(stdout);
}

void gemm_eval(
		const mtk::oztcecgemm::gemm_list_t& gemm_list
		) {
	mtk::oztcecgemm::handle_t oztcecgemm_handle;
	mtk::oztcecgemm::create(&oztcecgemm_handle);
	mtk::oztcecgemm::reallocate_working_memory(oztcecgemm_handle, gemm_list);

	std::size_t max_AB_count = 0;
	std::size_t max_C_size = 0;
	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		max_AB_count = std::max(max_AB_count, m * k + k * n);
		max_C_size  = std::max(max_C_size , m * n *
				mtk::oztcecgemm::get_data_size_in_byte(
				mtk::oztcecgemm::get_output_type(std::get<3>(gemm))));
	}

	auto mat_AB_uptr = cutf::memory::get_device_unique_ptr<float>(max_AB_count);
	auto mat_C_uptr  = cutf::memory::get_device_unique_ptr<std::uint8_t>(max_C_size);

	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_MT19937);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), 0));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), mat_AB_uptr.get(), max_AB_count));

	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		const auto mode = std::get<3>(gemm);
		gemm_eval_core(
				mtk::oztcecgemm::op_n,
				mtk::oztcecgemm::op_n,
				m, n, k,
				mat_AB_uptr.get(), m,
				mat_AB_uptr.get() + m * k, k,
				mat_C_uptr.get(), m,
				[&](
						const mtk::oztcecgemm::operation_t op_a,
						const mtk::oztcecgemm::operation_t op_b,
						const std::size_t m,
						const std::size_t n,
						const std::size_t k,
						const float* const a_ptr, const std::size_t lda,
						const float* const b_ptr, const std::size_t ldb,
						void* const c_ptr, const std::size_t ldc
									) {
					if (mtk::oztcecgemm::get_output_type(mode) == mtk::oztcecgemm::fp32) {
						using C_T = float;
						const C_T alpha = 1, beta = 0;
						mtk::oztcecgemm::gemm(
								oztcecgemm_handle,
								op_a, op_b,
								m, n, k,
								&alpha,
								a_ptr, lda,
								b_ptr, ldb,
								&beta,
								c_ptr, ldc,
								mode
								);
					} else {
						using C_T = double;
						const C_T alpha = 1, beta = 0;
						mtk::oztcecgemm::gemm(
								oztcecgemm_handle,
								op_a, op_b,
								m, n, k,
								&alpha,
								a_ptr, lda,
								b_ptr, ldb,
								&beta,
								c_ptr, ldc,
								mode
								);
					}
				},
				mode
				);
	}

	mtk::oztcecgemm::destroy(oztcecgemm_handle);
}

int main(int argc, char** argv) {
	mtk::oztcecgemm::gemm_list_t gemm_list;

	const std::vector<mtk::oztcecgemm::compute_mode_t> modes = {
		mtk::oztcecgemm::sgemm,
		mtk::oztcecgemm::fp32_split_3,
	};

	for (const auto mode : modes) {
		gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
					16,
					16,
					16,
					mode
					));
	}

	gemm_eval(gemm_list);
}
