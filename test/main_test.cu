#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <oztcecgemm/oztcecgemm.hpp>
#include <cutf/memory.hpp>
#include <cutf/hiprand.hpp>
#include <mateval/comparison_cuda.hpp>
#include <matfile/matfile.hpp>

constexpr unsigned test_count = 100;

constexpr unsigned long long seed = 0;

inline mtk::mateval::layout_t conv_layout_oz2mateval(const mtk::oztcecgemm::operation_t op) {
	if (op == mtk::oztcecgemm::op_n) {
		return mtk::mateval::col_major;
	}
	return mtk::mateval::row_major;
}

template <class T>
__global__ void adjust_urand_kernel(
		T* const ptr,
		const T min_urand,
		const T max_urand,
		const std::size_t n
		) {
	const auto tid = threadIdx.x + blockDim.x * blockIdx.x;
	if (tid >= n) {
		return;
	}

	const auto v = ptr[tid];
	ptr[tid] = v * (max_urand - min_urand) + min_urand;
}

template <class T>
void adjust_urand(
		T* const ptr,
		const T min_urand,
		const T max_urand,
		const std::size_t n
		) {
	const auto block_size = 256lu;
	const auto grid_size = (n + block_size - 1) / block_size;

	adjust_urand_kernel<T><<<grid_size, block_size>>>(
			ptr,
			min_urand, max_urand,
			n
			);
}

template <class C_T, class AB_T, class MATMUL_FUNC>
void gemm_eval_core(
		const mtk::oztcecgemm::operation_t op_a,
		const mtk::oztcecgemm::operation_t op_b,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const AB_T* const a_ptr, const std::size_t lda,
		const AB_T* const b_ptr, const std::size_t ldb,
		C_T* const c_ptr, const std::size_t ldc,
		const MATMUL_FUNC matmul_func,
		const mtk::oztcecgemm::compute_mode_t mode
		) {
	matmul_func(
			op_a, op_b,
			m, n, k,
			a_ptr, lda,
			b_ptr, ldb,
			c_ptr, ldc
			);

	mtk::mateval::error_map_t error;
	if (mtk::oztcecgemm::get_output_type(mode) == mtk::oztcecgemm::fp32) {
		error = mtk::mateval::cuda::get_error_AxB(
				mtk::mateval::relative_residual | mtk::mateval::max_relative_error,
				m, n, k,
				conv_layout_oz2mateval(op_a),
				conv_layout_oz2mateval(op_b),
				mtk::mateval::col_major,
				a_ptr, lda,
				b_ptr, ldb,
				reinterpret_cast<float*>(c_ptr), ldc
				);
	} else {
		error = mtk::mateval::cuda::get_error_AxB(
				mtk::mateval::relative_residual | mtk::mateval::max_relative_error,
				m, n, k,
				conv_layout_oz2mateval(op_a),
				conv_layout_oz2mateval(op_b),
				mtk::mateval::col_major,
				a_ptr, lda,
				b_ptr, ldb,
				reinterpret_cast<double*>(c_ptr), ldc
				);
	}

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto start_clock = std::chrono::system_clock::now();

	for (unsigned i = 0; i < test_count; i++) {
		matmul_func(
				op_a, op_b,
				m, n, k,
				a_ptr, lda,
				b_ptr, ldb,
				c_ptr, ldc
				);
	}

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();

	const auto elapsed_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_clock - start_clock).count() * 1e-9 / test_count;

	const auto throughput = 2 * m * n * k / elapsed_time;

	std::printf("%s,%lu,%lu,%lu,%e,%e,%e\n",
			mtk::oztcecgemm::get_compute_mode_name_str(mode).c_str(),
			m, n, k,
			error.at(mtk::mateval::relative_residual),
			error.at(mtk::mateval::max_relative_error),
			throughput * 1e-12
			);
	std::fflush(stdout);
}

template <class T>
void gemm_eval(
		const mtk::oztcecgemm::gemm_list_t& gemm_list,
		const std::string input_mode
		) {
	mtk::oztcecgemm::handle_t oztcecgemm_handle;
	mtk::oztcecgemm::create(&oztcecgemm_handle);
	mtk::oztcecgemm::reallocate_working_memory(oztcecgemm_handle, gemm_list);

	std::size_t max_AB_count = 0;
	std::size_t max_C_size = 0;
	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		max_AB_count = std::max(max_AB_count, m * k + k * n);
		max_C_size  = std::max(max_C_size , m * n *
				mtk::oztcecgemm::get_data_size_in_byte(
				mtk::oztcecgemm::get_output_type(std::get<3>(gemm))));
	}

	auto mat_AB_uptr = cutf::memory::get_device_unique_ptr<T>(max_AB_count);
	auto mat_C_uptr  = cutf::memory::get_device_unique_ptr<std::uint8_t>(max_C_size);

	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_MT19937);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), seed));
	if (input_mode == "normal01") {
		CUTF_CHECK_ERROR(cutf::hiprand::generate_normal(*cugen.get(), mat_AB_uptr.get(), max_AB_count, 0, 1));
	} else {
		CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), mat_AB_uptr.get(), max_AB_count));
	}

	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		const auto mode = std::get<3>(gemm);
		gemm_eval_core(
				mtk::oztcecgemm::op_n,
				mtk::oztcecgemm::op_n,
				m, n, k,
				mat_AB_uptr.get(), m,
				mat_AB_uptr.get() + m * k, k,
				mat_C_uptr.get(), m,
				[&](
						const mtk::oztcecgemm::operation_t op_a,
						const mtk::oztcecgemm::operation_t op_b,
						const std::size_t m,
						const std::size_t n,
						const std::size_t k,
						const T* const a_ptr, const std::size_t lda,
						const T* const b_ptr, const std::size_t ldb,
						void* const c_ptr, const std::size_t ldc
									) {
					if (mtk::oztcecgemm::get_output_type(mode) == mtk::oztcecgemm::fp32) {
						using C_T = float;
						const C_T alpha = 1, beta = 0;
						mtk::oztcecgemm::gemm(
								oztcecgemm_handle,
								op_a, op_b,
								m, n, k,
								&alpha,
								a_ptr, lda,
								b_ptr, ldb,
								&beta,
								c_ptr, ldc,
								mode
								);
					} else {
						using C_T = double;
						const C_T alpha = 1, beta = 0;
						mtk::oztcecgemm::gemm(
								oztcecgemm_handle,
								op_a, op_b,
								m, n, k,
								&alpha,
								a_ptr, lda,
								b_ptr, ldb,
								&beta,
								c_ptr, ldc,
								mode
								);
					}
				},
				mode
				);
	}

	mtk::oztcecgemm::destroy(oztcecgemm_handle);
}

template <class SRC_T, class DST_T>
__global__ void vector_copy_kernel(
		DST_T* const dst_ptr,
		const SRC_T* const src_ptr,
		const std::size_t N
		) {
	const auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid >= N) {
		return;
	}

	dst_ptr[tid] = src_ptr[tid];
}

template <class DEVICE_T>
void matfile_to_device_memory(
		DEVICE_T* const d_ptr,
		const std::string matfile_path
		) {
	std::size_t m, n;
	mtk::matfile::load_size(m, n, matfile_path);
	const auto dtype = mtk::matfile::load_dtype(matfile_path);

	auto h_mat_uptr = cutf::memory::get_host_unique_ptr<std::uint8_t>(m * n * mtk::matfile::get_dtype_size(dtype));

	mtk::matfile::load_dense(
			h_mat_uptr.get(),
			m,
			matfile_path
			);

	const std::size_t block_size = 256;
	const std::size_t grid_size = (m * n + block_size - 1) / block_size;

	if (dtype == mtk::matfile::fp32) {
		vector_copy_kernel<<<grid_size, block_size>>>(
				d_ptr,
				reinterpret_cast<float*>(h_mat_uptr.get()),
				m * n
				);
	} else if (dtype == mtk::matfile::fp64) {
		vector_copy_kernel<<<grid_size, block_size>>>(
				d_ptr,
				reinterpret_cast<double*>(h_mat_uptr.get()),
				m * n
				);
	} else {
		vector_copy_kernel<<<grid_size, block_size>>>(
				d_ptr,
				reinterpret_cast<long double*>(h_mat_uptr.get()),
				m * n
				);
	}
}

template <class T>
void gemm_eval_matfile(
		const mtk::oztcecgemm::gemm_list_t& gemm_list,
		const std::string matfile_A_path,
		const std::string matfile_B_path,
		const std::string matfile_C_path
		) {
	mtk::oztcecgemm::handle_t oztcecgemm_handle;
	mtk::oztcecgemm::create(&oztcecgemm_handle);
	mtk::oztcecgemm::reallocate_working_memory(oztcecgemm_handle, gemm_list);

	std::size_t max_AB_count = 0;
	std::size_t max_C_size = 0;
	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		max_AB_count = std::max(max_AB_count, m * k + k * n);
		max_C_size  = std::max(max_C_size , m * n *
				mtk::oztcecgemm::get_data_size_in_byte(
				mtk::oztcecgemm::get_output_type(std::get<3>(gemm))));
	}

	auto mat_AB_uptr = cutf::memory::get_device_unique_ptr<T>(max_AB_count);
	auto mat_C_uptr  = cutf::memory::get_device_unique_ptr<std::uint8_t>(max_C_size);
	auto mat_ref_uptr  = cutf::memory::get_host_unique_ptr<std::uint8_t>(max_C_size);


	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		const auto mode = std::get<3>(gemm);

		const auto a_ptr = mat_AB_uptr.get();
		const auto b_ptr = mat_AB_uptr.get() + m * k;
		const auto c_ptr = mat_C_uptr.get();

		matfile_to_device_memory(a_ptr, matfile_A_path);
		matfile_to_device_memory(b_ptr, matfile_B_path);
		mtk::matfile::load_dense(mat_ref_uptr.get(), m, matfile_C_path);

		mtk::mateval::error_map_t error;
		if (mtk::oztcecgemm::get_output_type(mode) == mtk::oztcecgemm::fp32) {
			using C_T = float;
			const C_T alpha = 1, beta = 0;
			mtk::oztcecgemm::gemm(
					oztcecgemm_handle,
					mtk::oztcecgemm::op_n, mtk::oztcecgemm::op_n,
					m, n, k,
					&alpha,
					a_ptr, m,
					b_ptr, k,
					&beta,
					c_ptr, m,
					mode
					);
			error = mtk::mateval::cuda::get_error(
					mtk::mateval::max_relative_error | mtk::mateval::relative_residual,
					m, n,
					mtk::mateval::col_major,
					mtk::mateval::col_major,
					c_ptr, m,
					reinterpret_cast<C_T*>(mat_ref_uptr.get()), m
					);
		} else {
			using C_T = double;
			const C_T alpha = 1, beta = 0;
			mtk::oztcecgemm::gemm(
					oztcecgemm_handle,
					mtk::oztcecgemm::op_n, mtk::oztcecgemm::op_n,
					m, n, k,
					&alpha,
					a_ptr, m,
					b_ptr, k,
					&beta,
					c_ptr, m,
					mode
					);
			error = mtk::mateval::cuda::get_error(
					mtk::mateval::max_relative_error | mtk::mateval::relative_residual,
					m, n,
					mtk::mateval::col_major,
					mtk::mateval::col_major,
					c_ptr, m,
					reinterpret_cast<C_T*>(mat_ref_uptr.get()), m
					);
		}

		std::printf("%s,%lu,%lu,%lu,%e,%e\n",
				mtk::oztcecgemm::get_compute_mode_name_str(mode).c_str(),
				m, n, k,
				error.at(mtk::mateval::relative_residual),
				error.at(mtk::mateval::max_relative_error)
				);
		std::fflush(stdout);
	}

	mtk::oztcecgemm::destroy(oztcecgemm_handle);
}

std::vector<mtk::oztcecgemm::compute_mode_t> get_supported_compute_mode() {
	return std::vector<mtk::oztcecgemm::compute_mode_t>{
		mtk::oztcecgemm::sgemm,
		mtk::oztcecgemm::dgemm,
		mtk::oztcecgemm::fp32_split_3,
		mtk::oztcecgemm::fp64_int8_6,
		mtk::oztcecgemm::fp64_int8_7,
		mtk::oztcecgemm::fp64_int8_8,
		mtk::oztcecgemm::fp64_int8_9,
	};
}

std::vector<mtk::oztcecgemm::compute_mode_t> get_compute_mode_list_from_argv(
		const std::size_t count,
		char** argv
		) {
	std::vector<mtk::oztcecgemm::compute_mode_t> mode_list;

	for (std::size_t i = 0; i < count; i++) {
		bool added = false;
		for (const auto m : get_supported_compute_mode()) {
			if (std::string(argv[i]) == mtk::oztcecgemm::get_compute_mode_name_str(m)) {
				mode_list.push_back(m);
				added = true;
				break;
			}
		}
		if (!added) {
			std::fprintf(stderr, "Warning: Unknown compute mode \"%s\"\n", argv[i]);
		}
	}

	return mode_list;
}

void print_usage(
		const char* const program_name
		) {
	std::string compute_mode_list_str = "";
	for (const auto& name : get_supported_compute_mode()) {
		compute_mode_list_str += mtk::oztcecgemm::get_compute_mode_name_str(name) + " ";
	}

	std::printf(
			"Usage:\n"
			"%s matfile [/path/to/A.matrix] [/path/to/B.matrix] [/path/to/Ref.matrix] [Computing mode list]\n"
			"%s [urand01|normal01] [seq|exp2] [start_N] [end_N] [interval_N] [Computing mode list]\n"
			"Compute modes:\n"
			" %s\n",
			program_name,
			program_name,
			compute_mode_list_str.c_str()
			);
}

int main(int argc, char** argv) {

	if (argc <= 2) {
		print_usage(argv[0]);
		return 1;
	}

	const auto input_mode = std::string(argv[1]);
	if (input_mode == "matfile") {
		if (argc <= 5) {
			print_usage(argv[0]);
			return 1;
		}

		const auto matfile_A_path = std::string(argv[2]);
		const auto matfile_B_path = std::string(argv[3]);
		const auto matfile_C_path = std::string(argv[4]);
		const auto compute_mode_list = get_compute_mode_list_from_argv(argc - 5, argv + 5);

		std::size_t am, an, bm, bn, cm, cn;
		mtk::matfile::load_size(am, an, matfile_A_path);
		mtk::matfile::load_size(bm, bn, matfile_B_path);
		mtk::matfile::load_size(cm, cn, matfile_C_path);
		if (am != cm || bn != cn || an != bm) {
			std::fprintf(stderr, "Error: matrix shapes are mismatch: A=(%lu, %lu), B=(%lu, %lu), C=(%lu, %lu)\n",
					am, an,
					bm, bn,
					cm, cn
					);
			return 1;
		}

		mtk::oztcecgemm::gemm_list_t fp32in_gemm_list;
		mtk::oztcecgemm::gemm_list_t fp64in_gemm_list;

		for (auto compute_mode : compute_mode_list) {
			if (mtk::oztcecgemm::get_output_type(compute_mode) == mtk::oztcecgemm::fp32) {
				fp32in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
							cm,
							cn,
							an,
							compute_mode
							));
			} else {
				fp64in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
							cm,
							cn,
							an,
							compute_mode
							));
			}
		}

		std::printf("mode,m,n,k,residual,max_relative,throughput_in_tflops\n");
		std::fflush(stdout);
		if (fp32in_gemm_list.size() != 0) {
			gemm_eval<float>(fp32in_gemm_list, input_mode);
		}
		if (fp64in_gemm_list.size() != 0) {
			gemm_eval<double>(fp64in_gemm_list, input_mode);
		}

	} else if (input_mode == "urand01" || input_mode == "normal01") {
		if (argc <= 6) {
			print_usage(argv[0]);
			return 1;
		}
		const auto N_mode = std::string(argv[2]);
		if (N_mode != "seq" && N_mode != "exp2") {
			std::fprintf(stderr, "Error: unknown N mode \"%s\"\n", N_mode.c_str());
			return 1;
		}
		const auto min_N = std::stoul(argv[3]);
		const auto max_N = std::stoul(argv[4]);
		const auto interval_N = std::stoul(argv[5]);
		const auto compute_mode_list = get_compute_mode_list_from_argv(argc - 6, argv + 6);

		mtk::oztcecgemm::gemm_list_t fp32in_gemm_list;
		mtk::oztcecgemm::gemm_list_t fp64in_gemm_list;

		for (std::size_t N = min_N; N <= max_N; N += interval_N) {
			auto real_N = N;
			if (N_mode == "exp2") {real_N = 1lu << N;}

			for (auto compute_mode : compute_mode_list) {
				if (mtk::oztcecgemm::get_output_type(compute_mode) == mtk::oztcecgemm::fp32) {
					fp32in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
								real_N,
								real_N,
								real_N,
								compute_mode
								));
				} else {
					fp64in_gemm_list.push_back(std::tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
								real_N,
								real_N,
								real_N,
								compute_mode
								));
				}
			}
		}

		std::printf("mode,m,n,k,residual,max_relative,throughput_in_tflops\n");
		std::fflush(stdout);
		if (fp32in_gemm_list.size() != 0) {
			gemm_eval<float>(fp32in_gemm_list, input_mode);
		}
		if (fp64in_gemm_list.size() != 0) {
			gemm_eval<double>(fp64in_gemm_list, input_mode);
		}
	} else {
		std::fprintf(stderr, "Error: Unknown input mode \"%s\"\n", input_mode.c_str());
		return 1;
	}
}
