#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <oztcecgemm/oztcecgemm.hpp>
#include <cutf/memory.hpp>
#include <cutf/hiprand.hpp>
#include <mateval/comparison_cuda.hpp>

constexpr unsigned test_count = 100;

inline mtk::mateval::layout_t conv_layout_oz2mateval(const mtk::oztcecgemm::operation_t op) {
	if (op == mtk::oztcecgemm::op_n) {
		return mtk::mateval::col_major;
	}
	return mtk::mateval::row_major;
}

template <class T, class MATMUL_FUNC>
void gemm_eval_core(
		const mtk::oztcecgemm::operation_t op_a,
		const mtk::oztcecgemm::operation_t op_b,
		const std::size_t m,
		const std::size_t n,
		const std::size_t k,
		const T* const a_ptr, const std::size_t lda,
		const T* const b_ptr, const std::size_t ldb,
		T* const c_ptr, const std::size_t ldc,
		const MATMUL_FUNC matmul_func,
		const std::string mode_name
		) {
	matmul_func(
			op_a, op_b,
			m, n, k,
			a_ptr, lda,
			b_ptr, ldb,
			c_ptr, ldc
			);

	const auto error = mtk::mateval::cuda::get_error_AxB(
			mtk::mateval::relative_residual | mtk::mateval::max_relative_error,
			m, n, k,
			conv_layout_oz2mateval(op_a),
			conv_layout_oz2mateval(op_b),
			mtk::mateval::col_major,
			a_ptr, lda,
			b_ptr, ldb,
			c_ptr, ldc
			);

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto start_clock = std::chrono::system_clock::now();

	for (unsigned i = 0; i < test_count; i++) {
		matmul_func(
				op_a, op_b,
				m, n, k,
				a_ptr, lda,
				b_ptr, ldb,
				c_ptr, ldc
				);
	}

	CUTF_CHECK_ERROR(hipDeviceSynchronize());
	const auto end_clock = std::chrono::system_clock::now();

	const auto elapsed_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end_clock - start_clock).count() * 1e-9 / test_count;

	const auto throughput = 2 * m * n * k / elapsed_time;

	std::printf("%s,%lu,%lu,%lu,%e,%e,%e\n",
			mode_name.c_str(),
			m, n, k,
			error.at(mtk::mateval::relative_residual),
			error.at(mtk::mateval::max_relative_error),
			throughput * 1e-12
			);
	std::fflush(stdout);
}

template <class T>
void gemm_eval(
		const mtk::oztcecgemm::gemm_list_t& gemm_list
		) {
	mtk::oztcecgemm::handle_t oztcecgemm_handle;
	mtk::oztcecgemm::create(&oztcecgemm_handle);
	mtk::oztcecgemm::reallocate_working_memory(oztcecgemm_handle, gemm_list);

	std::size_t max_AB_size = 0;
	std::size_t max_C_size = 0;
	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		max_AB_size = std::max(max_AB_size, m * k + k * n);
		max_C_size  = std::max(max_C_size , m * n);
	}

	auto mat_AB_uptr = cutf::memory::get_device_unique_ptr<T>(max_AB_size);
	auto mat_C_uptr  = cutf::memory::get_device_unique_ptr<T>(max_C_size);

	auto cugen = cutf::hiprand::get_curand_unique_ptr(HIPRAND_RNG_PSEUDO_MT19937);
	CUTF_CHECK_ERROR(hiprandSetPseudoRandomGeneratorSeed(*cugen.get(), 0));
	CUTF_CHECK_ERROR(cutf::hiprand::generate_uniform(*cugen.get(), mat_AB_uptr.get(), max_AB_size));

	for (const auto gemm : gemm_list) {
		const auto m = std::get<0>(gemm);
		const auto n = std::get<1>(gemm);
		const auto k = std::get<2>(gemm);
		const auto mode = std::get<3>(gemm);
		gemm_eval_core(
				mtk::oztcecgemm::op_n,
				mtk::oztcecgemm::op_n,
				m, n, k,
				mat_AB_uptr.get(), m,
				mat_AB_uptr.get() + m * k, k,
				mat_C_uptr.get(), m,
				[&](
						const mtk::oztcecgemm::operation_t op_a,
						const mtk::oztcecgemm::operation_t op_b,
						const std::size_t m,
						const std::size_t n,
						const std::size_t k,
						const T* const a_ptr, const std::size_t lda,
						const T* const b_ptr, const std::size_t ldb,
						T* const c_ptr, const std::size_t ldc
									) {
					const T alpha = 1, beta = 0;
					mtk::oztcecgemm::gemm(
							oztcecgemm_handle,
							op_a, op_b,
							m, n, k,
							&alpha,
							a_ptr, lda,
							b_ptr, ldb,
							&beta,
							c_ptr, ldc,
							mode
							);
				},
				mtk::oztcecgemm::get_compute_mode_name_str(mode)
				);
	}

	mtk::oztcecgemm::destroy(oztcecgemm_handle);
}

int main(int argc, char** argv) {
	mtk::oztcecgemm::gemm_list_t gemm_list;

	gemm_list.push_back(std::make_tuple<std::size_t, std::size_t, std::size_t, mtk::oztcecgemm::compute_mode_t>(
				1024,
				1024,
				1024,
				mtk::oztcecgemm::fp32_split_3
				));

	gemm_eval<float>(gemm_list);
}
